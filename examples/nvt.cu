#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2017. A short range forces example.

This file contains a good example of how UAMMD works and how to configure and launch a simulation.

It describes a LJ liquid simulation in a periodic box.
Two types of LJ particles exist, starting in a random configuration.
Each type is attracted to a different z plane.
This makes the particles form a kind of pillars going from one type to the other.

Commented code is available to show other available features.
  
Needs cli input arguments with a system size, etc, look for "argv"

Or just run: ./a.out 20 128 0.01 1.0 500 -1 1.0
for a quick test
You should get ~ 90 FPS on a GTX 980

You can visualize the reuslts with superpunto

  
 */

//This include contains the basic needs for an uammd project
#include"uammd.cuh"
//The rest can be included depending on the used modules
#include"Integrator/VerletNVT.cuh"
#include"Interactor/NeighbourList/CellList.cuh"
#include"Interactor/PairForces.cuh"
#include"Interactor/Potential/Potential.cuh"
#include"utils/InitialConditions.cuh"
#include<fstream>


using namespace uammd;
using namespace std;

int main(int argc, char *argv[]){

  if(argc<8){
    std::cerr<<"ERROR, I need some parameters!!\nTry to run me with:\n./a.out 20 128 0.01 1.0 500 -1 1.0"<<std::endl;
    exit(1);
  }
  int N = pow(2,atoi(argv[1]));//atoi(argv[1]));

  //UAMMD System entity holds information about the GPU and tools to interact with the computer itself (such as a loging system). All modules need a System to work on.
  
  auto sys = make_shared<System>();

  //Modules will ask System when they need a random number (i.e for seeding the GPU RNG).
  ullint seed = 0xf31337Bada55D00dULL^time(NULL);
  sys->rng().setSeed(seed);

  //ParticleData stores all the needed properties the simulation will need.
  //Needs to start with a certain number of particles, which can be changed mid-simulation
  //If UAMMD is to be used as a plugin for other enviroment or custom code, ParticleData should accept references to
  // properties allocated and handled by the user, this is a non-implemented work in progress as of now though.
  auto pd = make_shared<ParticleData>(N, sys);

  //Some modules need a simulation box (i.e PairForces for the PBC)
  Box box(std::stod(argv[2]));//std::stod(argv[2]));
  //Initial positions
  {
    //Ask pd for a property like so:
    auto pos = pd->getPos(access::location::cpu, access::mode::write);    

    auto initial =  initLattice(box.boxSize*std::stod(argv[4]), N, fcc);    

    fori(0,N){
      pos.raw()[i] = initial[i];
      //Type of particle is stored in .w
      pos.raw()[i].w = 0;
    }    
  }
  

  //Modules can work on a certain subset of particles if needed, the particles can be grouped following any criteria
  //The builtin ones will generally work faster than a custom one. See ParticleGroup.cuh for a list
  
  //A group created with no criteria will contain all the particles  
  auto pg = make_shared<ParticleGroup>(pd, sys, "All");  
  
  ofstream out("kk");
  
   // {
   //   auto pos = pd->getPos(access::location::cpu, access::mode::read);
   //   out<<"#"<<endl;
   //   fori(0,N){
   //     out<<pos.raw()[i]<<endl;
   //   }
   // }

  //Some modules need additional parameters, in this case VerletNVT needs dt, temperature...
  //When additional parameters are needed, they need to be supplied in a form similar to this:

  using Verlet = VerletNVT::GronbechJensen;
  Verlet::Parameters par;
  par.temperature = std::stod(argv[7]);
  par.dt = std::stod(argv[3]);
  par.viscosity = 1.0/(6*M_PI);  
  auto verlet = make_shared<Verlet>(pd, pg, sys, par);


  //Modules working with pairs of particles usually ask for a Potential object
  //PairForces decides if it should use a neighbour list or treat the system as NBody,
  //You can force the use of a certain neighbour list passing its name as a second template argument

  using PairForces = PairForces<Potential::LJ>;

  //This is the general interface for setting up a potential
  auto pot = make_shared<Potential::LJ>(sys);
  {
    //Each Potential describes the pair interactions with certain parameters.
    //The needed ones are in InputPairParameters inside each potential, in this case:
    Potential::LJ::InputPairParameters par;
    par.epsilon = 1.0;
    par.shift = false;

    par.sigma = 1;
    par.cutOff = 2.5*par.sigma;
    //Once the InputPairParameters has been filled accordingly for a given pair of types,
    //a potential can be informed like this:
    pot->setPotParameters(0, 0, par);
  }

  PairForces::Parameters params;
  params.box = box;  //Box to work on
  auto pairforces = make_shared<PairForces>(pd, pg, sys, params, pot);
  
  //You can add as many modules as necessary
  verlet->addInteractor(pairforces);


  //You can issue a logging event like this, a wide variety of log levels exists (see System.cuh).
  //A maximum log level is set in System.cuh, every logging event with a level superior to the max will result in
  // absolutely no overhead, so dont be afraid to write System::DEBUGX log calls.
  sys->log<System::MESSAGE>("RUNNING!!!");

  //Ask ParticleData to sort the particles in memory!
  //It is a good idea to sort the particles once in a while during the simulation
  //This can increase performance considerably as it improves coalescence.
  //Sorting the particles will cause the particle arrays to change in order and (possibly) address.
  //This changes will be informed with signals and any module that needs to be aware of such changes
  //will acknowedge it through a callback (see ParticleData.cuh).
  pd->sortParticles();
        
  Timer tim;
  tim.tic();
  int nsteps = std::atoi(argv[5]);
  int printSteps = std::atoi(argv[6]);
  ofstream velout("vel.dat");
  //Run the simulation
  forj(0,nsteps){
    //This will instruct the integrator to take the simulation to the next time step,
    //whatever that may mean for the particular integrator (i.e compute forces and update positions once)
    verlet->forwardTime();

    //Write results
    if(printSteps > 0 && j%printSteps==0)
    {
      sys->log<System::DEBUG1>("[System] Writing to disk...");
      //continue;
      auto pos = pd->getPos(access::location::cpu, access::mode::read);
      auto vel = pd->getVel(access::location::cpu, access::mode::read);
      const int * sortedIndex = pd->getIdOrderedIndices(access::location::cpu);
      out<<"#Lx="<<0.5*box.boxSize.x<<";Ly="<<0.5*box.boxSize.y<<";Lz="<<0.5*box.boxSize.z<<";"<<endl;
      real3 p;
      velout<<"#"<<endl;
      fori(0,N){
	real4 pc = pos.raw()[sortedIndex[i]];
	p =  box.apply_pbc(make_real3(pc));
	int type = pc.w;
	out<<p<<" "<<0.5*(type==1?2:1)<<" "<<type<<endl;
	velout<<vel.raw()[i].x<<" "<<vel.raw()[i].y<<" "<<vel.raw()[i].z<<"\n";
      }
      velout<<flush;

    }    
    //Sort the particles every few steps
    //It is not an expensive thing to do really.
    if(j%500 == 0){
      pd->sortParticles();
    }
  }
  
  auto totalTime = tim.toc();
  sys->log<System::MESSAGE>("mean FPS: %.2f", nsteps/totalTime);
  //sys->finish() will ensure a smooth termination of any UAMMD module.
  sys->finish();

  return 0;
}