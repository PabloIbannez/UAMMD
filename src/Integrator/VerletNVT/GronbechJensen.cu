#include "hip/hip_runtime.h"

/*Raul P. Pelaez 2017. Verlet NVT Integrator module.

  This module integrates the dynamic of the particles using a two step velocity verlet MD algorithm
  that conserves the temperature, volume and number of particles.
  
  The algorithm implemented is GronbechJensen[1]
 Usage:
 
    Create the module as any other integrator with the following parameters:
    
    
    auto sys = make_shared<System>();
    auto pd = make_shared<ParticleData>(N,sys);
    auto pg = make_shared<ParticleGroup>(pd,sys, "All");
    
    using NVT = VerletNVT::GronbechJensen;
    NVT::Parameters par;
     par.temperature = 1.0;
     par.dt = 0.01;
     par.damping = 1.0;
     par.is2D = false;

    auto verlet = make_shared<NVT>(pd, pg, sys, par);
      
    //Add any interactor
    verlet->addInteractor(...);
    ...
    
    //forward simulation 1 dt:
    
    verlet->forwardTime();
    
-----
References:

[1] N. Gronbech-Jensen, and O. Farago: "A simple and effective Verlet-type
algorithm for simulating Langevin dynamics", Molecular Physics (2013).
http://dx.doi.org/10.1080/00268976.2012.760055 

 */

#include"../VerletNVT.cuh"

#ifndef SINGLE_PRECISION
#define hiprandGenerateNormal hiprandGenerateNormalDouble
#endif


namespace uammd{
  namespace VerletNVT{
    namespace GronbechJensen_ns{

      //Integrate the movement 1 dt and reset the forces in the first step
      template<int step>
      __global__ void integrateGPU(real4 __restrict__  *pos,
				   real3 __restrict__ *vel,
				   real4 __restrict__  *force,
				   const real __restrict__ *mass,
				   const real __restrict__ *radius,				   
				   const real3 __restrict__ *noise,
				   ParticleGroup::IndexIterator indexIterator,
				   int N,
				   real dt, real viscosity, bool is2D){
	const int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=N) return;
	//Index of current particle in group
	const int i = indexIterator[id];       


	real invMass = real(1.0);
	if(mass){
	  invMass = real(1.0)/mass[i];
	}
	real radius_i = real(1.0);
	if(radius){
	  radius_i = radius[i];
	}
	
	const real damping = real(6.0)*real(M_PI)*viscosity*radius_i;

	if(step==1){
	  real b = real(1.0)/(real(1.0) + damping*dt*invMass*real(0.5));
	
	  real a = (real(1.0)-damping*dt*real(0.5)*invMass)*b;
       
	
	  real3 p = make_real3(pos[i]);
	  p = p +
	    b*dt*vel[i] +
	    b*dt*dt*real(0.5)*invMass*make_real3(force[i]) +
	    b*dt*real(0.5)*invMass*sqrtf(2.0f*radius_i)*noise[id];
	
	  pos[i] = make_real4(p, pos[i].w);

	  vel[i] = a*vel[i] +
	    dt*real(0.5)*invMass*a*make_real3(force[i]) +
	    b*invMass*sqrtf(2.0f*radius_i)*noise[id];
	  
	  if(is2D) vel[i].z = real(0.0);
	  
	  force[i] = make_real4(0);
	}      
	else{
	  vel[i] += dt*real(0.5)*invMass*make_real3(force[i]);
	}

      }


    }    
      //Move the particles in my group 1 dt in time.
    void GronbechJensen::forwardTime(){
      for(auto forceComp: interactors) forceComp->updateSimulationTime(steps*dt);
    
      steps++;
      sys->log<System::DEBUG1>("[%s] Performing integration step %d", name.c_str(), steps);
    
      int numberParticles = pg->getNumberParticles();
      //Handle if the number of particles in my group has changed
      if(noise.size() != numberParticles)  noise.resize(numberParticles);

    
      int Nthreads=128;
      int Nblocks=numberParticles/Nthreads + ((numberParticles%Nthreads)?1:0);

    
      //First simulation step is special
      if(steps==1){
	{
	  auto groupIterator = pg->getIndexIterator(access::location::gpu);
	  auto force = pd->getForce(access::location::gpu, access::mode::write);     
	  fillWithGPU<<<Nblocks, Nthreads>>>(force.raw(), groupIterator, make_real4(0), numberParticles);
	}
	for(auto forceComp: interactors){
	  forceComp->updateTemperature(temperature);
	  forceComp->updateTimeStep(dt);
	  forceComp->sumForce(forceStream);
	}
	/*Gen noise*/
	genNoise(stream);
	hipDeviceSynchronize();
      }
      genNoise(stream);
      //First integration step
      {

	//An iterator with the global indices of my groups particles
	auto groupIterator = pg->getIndexIterator(access::location::gpu);
	//Get all necessary properties
	auto pos = pd->getPos(access::location::gpu, access::mode::readwrite);
	auto vel = pd->getVel(access::location::gpu, access::mode::readwrite);
	auto force = pd->getForce(access::location::gpu, access::mode::read);     
	//Mass is assumed 1 for all particles if it has not been set.
	auto mass = pd->getMassIfAllocated(access::location::gpu, access::mode::read);
	auto radius = pd->getRadiusIfAllocated(access::location::gpu, access::mode::read);
	//Second half of noise vector is used for first integration step
	auto noise_ptr = thrust::raw_pointer_cast(noise.data());
      
	/*First step integration and reset forces*/

	GronbechJensen_ns::integrateGPU<1><<<Nblocks, Nthreads, 0, stream>>>(pos.raw(),
								    vel.raw(),
								    force.raw(),
								    mass.raw(),
								    radius.raw(),
								    noise_ptr,
								    groupIterator,
								    numberParticles, dt, viscosity, is2D);
      }      
      
      //Gen noise and compute forces at the same time
      hipEventRecord(forceEvent, stream);
      //Gen noise for two integration steps at once
      genNoise(stream);
      //Compute all the forces
      hipStreamWaitEvent(forceStream, forceEvent, 0);
      for(auto forceComp: interactors) forceComp->sumForce(forceStream);
      hipEventRecord(forceEvent, forceStream);
    
      //Second integration step
      {
	auto groupIterator = pg->getIndexIterator(access::location::gpu);
      
	auto pos = pd->getPos(access::location::gpu, access::mode::readwrite);
	auto vel = pd->getVel(access::location::gpu, access::mode::readwrite);
	auto force = pd->getForce(access::location::gpu, access::mode::read);
      
	auto noise_ptr = thrust::raw_pointer_cast(noise.data());
      
	auto mass = pd->getMassIfAllocated(access::location::gpu, access::mode::read);
	auto radius = pd->getRadiusIfAllocated(access::location::gpu, access::mode::read);

      	//Wait untill all forces have been summed
	hipStreamWaitEvent(stream, forceEvent, 0);
	GronbechJensen_ns::integrateGPU<2><<<Nblocks, Nthreads, 0 , stream>>>(pos.raw(),
						vel.raw(),
						force.raw(),
						mass.raw(),
						radius.raw(),
						noise_ptr,
						groupIterator,
						numberParticles, dt, viscosity, is2D);      
      }

    }
  }
}
