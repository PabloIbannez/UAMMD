#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2017. Verlet NVT Integrator module.

  This module integrates the dynamic of the particles using a two step velocity verlet MD algorithm
  that conserves the temperature, volume and number of particles.

  For that several thermostats are (should be, currently only one) implemented:

    -Velocity damping and gaussian noise 
    - BBK ( TODO)
    - SPV( TODO)
 Usage:
 
    Create the module as any other integrator with the following parameters:
    
    
    auto sys = make_shared<System>();
    auto pd = make_shared<ParticleData>(N,sys);
    auto pg = make_shared<ParticleGroup>(pd,sys, "All");
    
    
    VerletNVT::Parameters par;
     par.temperature = 1.0;
     par.dt = 0.01;
     par.damping = 1.0;
     par.is2D = false;

    auto verlet = make_shared<VerletNVT>(pd, pg, sys, par);
      
    //Add any interactor
    verlet->addInteractor(...);
    ...
    
    //forward simulation 1 dt:
    
    verlet->forwardTime();
    
TODO:

100- Outsource thermostat logic to a functor (external or internal)
100-Implement thermostat from https://arxiv.org/pdf/1212.1244.pdf
 */

#include"VerletNVT.cuh"


#ifndef SINGLE_PRECISION
#define hiprandGenerateNormal hiprandGenerateNormalDouble
#endif

namespace uammd{


  namespace VerletNVT_ns{
    //Fill the initial velocities of the particles in my group with a gaussian distribution according with my temperature.
    __global__ void initialVelocities(real3* vel, const real* mass, const real3* noise,
				      ParticleGroup::IndexIterator indexIterator, //global index of particles in my group
				      real vamp, bool is2D, int N){
      int id = blockIdx.x*blockDim.x + threadIdx.x;      
      if(id>=N) return;
      int i = indexIterator[id];
      
      real mass_i = real(1.0);
      if(mass) mass_i = mass[i];
      int index = indexIterator[i];
      vel[index].x = vamp*noise[i].x/mass_i;
      vel[index].y = vamp*noise[i].y/mass_i;
      if(!is2D){
	vel[index].z = vamp*noise[i].z/mass_i;
      }
    }
    
  }
  
  VerletNVT::VerletNVT(shared_ptr<ParticleData> pd,
		       shared_ptr<ParticleGroup> pg,
		       shared_ptr<System> sys,		       
		       VerletNVT::Parameters par):
    Integrator(pd, pg, sys, "VerletNVT"),
    dt(par.dt), temperature(par.temperature), damping(par.damping), is2D(par.is2D),
    steps(0){
    
    sys->log<System::MESSAGE>("[VerletNVT] Temperature: %.3f", temperature);
    sys->log<System::MESSAGE>("[VerletNVT] Time step: %.3f", dt);
    sys->log<System::MESSAGE>("[VerletNVT] Damping constant: %.3f", damping);
    if(is2D){
      sys->log<System::MESSAGE>("[VerletNVT] Working in 2D mode.");
    }

    this->noiseAmplitude = sqrt(dt*damping*temperature);

    //Init rng
    hiprandCreateGenerator(&curng, HIPRAND_RNG_PSEUDO_DEFAULT);
    
    hiprandSetPseudoRandomGeneratorSeed(curng, sys->rng().next());
    
    int numberParticles = pg->getNumberParticles();
    noise.resize(2*numberParticles);

    int Nthreads=128;
    int Nblocks=numberParticles/Nthreads + ((numberParticles%Nthreads)?1:0);

           
    //This shit is obscure, hiprand will only work with an even number of elements
    real* noise_ptr = (real *) thrust::raw_pointer_cast(noise.data());
    //Warm cuRNG
    hiprandGenerateNormal(curng, noise_ptr, 3*noise.size(), 0.0, 1.0);
    hiprandGenerateNormal(curng, noise_ptr, 3*noise.size(), 0.0, 1.0);

    if(pd->isVelAllocated()){
      sys->log<System::WARNING>("[VerletNVT] Velocity will be overwritten to ensure temperature conservation!");
    }
    {
      auto vel_handle = pd->getVel(access::location::gpu, access::mode::write);
      auto groupIterator = pg->getIndexIterator(access::location::gpu);
      
      real velAmplitude = sqrt(3.0*temperature);
      
      auto noise_ptr = thrust::raw_pointer_cast(noise.data());
      real * mass_ptr = nullptr;
      if(pd->isMassAllocated()){
	auto mass = pd->getMass(access::location::gpu, access::mode::read);
	mass_ptr = mass.raw();
      }
      
      VerletNVT_ns::initialVelocities<<<Nblocks, Nthreads>>>(vel_handle.raw(),
							     mass_ptr,
							     noise_ptr,
							     groupIterator,
							     velAmplitude, is2D, numberParticles);
      hiprandGenerateNormal(curng, (real*)noise_ptr, 3*numberParticles + ((3*numberParticles)%2), 0.0, 1.0);
      
    }

    hipStreamCreate(&stream);
    hipStreamCreate(&forceStream);
    hipEventCreate(&forceEvent);
    //This line makes the code go much slower, I do not know why    
    //hipEventCreateWithFlags(&forceEvent, hipEventDisableTiming);
  }


  
  VerletNVT::~VerletNVT(){
    hiprandDestroyGenerator(curng);
    hipStreamDestroy(stream);
    hipEventDestroy(forceEvent);
  }



  namespace VerletNVT_ns{

    //Integrate the movement 1 dt and reset the forces in the first step
    template<int step>
      __global__ void integrateGPU(real4 __restrict__  *pos,
				   real3 __restrict__ *vel,
				   real4 __restrict__  *force,
				   const real __restrict__ *mass,
				   const real3 __restrict__ *noise,
				   ParticleGroup::IndexIterator indexIterator,
				   int N,
				   real dt, real damping, bool is2D){
      const int id = blockIdx.x*blockDim.x+threadIdx.x;
      if(id>=N) return;
      //Index of current particle in group
      const int i = indexIterator[id];
	
      //Half step velocity
      //real3 oldVel = make_real3(vel[i]);
      //real3 newVel = oldVel + (make_real3(force[i])-damping*oldVel)*dt*real(0.5) + noise[id];
      real invMass = real(1.0);
      if(mass){
	invMass = real(1.0)/mass[i];
      }
      vel[i] += (make_real3(force[i])*invMass-damping*vel[i])*dt*real(0.5) + noise[id]*sqrtf(invMass);
      if(is2D) vel[i].z = real(0.0);

      //In the first step, upload positions
      if(step==1){
	//vel[i] = (1-dt*damping*real(0.5))*vel[i]-dt*real(0.5)*make_real3(force[i]) + noise[id];
	  

	const real3 newPos = make_real3(pos[i]) + vel[i]*dt;
	pos[i] = make_real4(newPos, pos[i].w);
	//Reset force
	force[i] = make_real4(0);
      }
      // else{
      //   vel[i] = ( vel[i] - make_real3(force[i])*dt*real(0.5) + noise[id]*sqrtMass) /(1+dt*damping*real(0.5));

	// }

      }


  }    
  

  //Fill noise array with a gaussian distribution with mean 0 and std noiseAmplitude
  void VerletNVT::genNoise(hipStream_t st){

    real * noise_ptr = (real *) thrust::raw_pointer_cast(noise.data());
    hiprandSetStream(curng, st);
    hiprandGenerateNormal(curng, (real*) noise_ptr,
			 3*noise.size(),
			 real(0.0), noiseAmplitude);
  }
  
  //Move the particles in my group 1 dt in time.
  void VerletNVT::forwardTime(){
    steps++;
    sys->log<System::DEBUG1>("[VerletNVT] Performing integration step %d", steps);
    
    int numberParticles = pg->getNumberParticles();
    //Handle if the number of particles in my group has changed
    if(noise.size() != 2*numberParticles)  noise.resize(2*numberParticles);

    
    int Nthreads=128;
    int Nblocks=numberParticles/Nthreads + ((numberParticles%Nthreads)?1:0);

    
    //First simulation step is special
    if(steps==1){
      {
	auto groupIterator = pg->getIndexIterator(access::location::gpu);
	auto force = pd->getForce(access::location::gpu, access::mode::write);     
	fillWithGPU<<<Nblocks, Nthreads>>>(force.raw(), groupIterator, make_real4(0), numberParticles);
      }
      for(auto forceComp: interactors) forceComp->sumForce(forceStream);
      /*Gen noise*/
      genNoise(stream);
      hipDeviceSynchronize();
    }
    
    //First integration step
    {

      //An iterator with the global indices of my groups particles
      auto groupIterator = pg->getIndexIterator(access::location::gpu);
      //Get all necessary properties
      auto pos = pd->getPos(access::location::gpu, access::mode::readwrite);
      auto vel = pd->getVel(access::location::gpu, access::mode::readwrite);
      auto force = pd->getForce(access::location::gpu, access::mode::read);     
      //Mass is assumed 1 for all particles if it has not been set.
      real * mass_ptr = nullptr;
      if(pd->isMassAllocated()){
	auto mass = pd->getMass(access::location::gpu, access::mode::read);
	mass_ptr = mass.raw();
      }
      //Second half of noise vector is used for first integration step
      auto noise_ptr = thrust::raw_pointer_cast(noise.data()) + numberParticles;
      
      /*First step integration and reset forces*/

      VerletNVT_ns::integrateGPU<1><<<Nblocks, Nthreads, 0, stream>>>(pos.raw(),
								      vel.raw(),
								      force.raw(),
								      mass_ptr,
								      noise_ptr,
								      groupIterator,
								      numberParticles, dt, damping, is2D);
    }
    //Gen noise and compute forces at the same time
    hipEventRecord(forceEvent, stream);
    //Gen noise for two integration steps at once
    genNoise(stream);
    //Compute all the forces
    hipStreamWaitEvent(forceStream, forceEvent, 0);
    for(auto forceComp: interactors) forceComp->sumForce(forceStream);
    hipEventRecord(forceEvent, forceStream);
    
    //Second integration step
    {
      auto groupIterator = pg->getIndexIterator(access::location::gpu);
      
      auto pos = pd->getPos(access::location::gpu, access::mode::readwrite);
      auto vel = pd->getVel(access::location::gpu, access::mode::readwrite);
      auto force = pd->getForce(access::location::gpu, access::mode::read);
      
      auto noise_ptr = thrust::raw_pointer_cast(noise.data());
      
      real * mass_ptr = nullptr;
      if(pd->isMassAllocated()){
	auto mass = pd->getMass(access::location::gpu, access::mode::read);
	mass_ptr = mass.raw();
      }
      //Wait untill all forces have been summed
      hipStreamWaitEvent(stream, forceEvent, 0);
      VerletNVT_ns::integrateGPU<2><<<Nblocks, Nthreads, 0 , stream>>>(pos.raw(),
								       vel.raw(),
								       force.raw(),
								       mass_ptr,
								       noise_ptr,
								       groupIterator,
								       numberParticles, dt, damping, is2D);      
    }

  }
  
}






































