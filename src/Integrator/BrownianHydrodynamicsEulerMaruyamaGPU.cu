#include "hip/hip_runtime.h"

/*Raul P. Pelaez 2016. Brownian Euler Maruyama with hydrodynamics integrator GPU kernels and callers

  Functions to integrate movement. The integration is done via a functor wich accesor ()
  takes a thrust::Tuple containing positions, velocities and forces on each particle. 

  TODO:
  100-Optimize, see .cpp
*/

#include"utils/hip/hip_vector_types.h"
#include"utils/helper_gpu.cuh"
#include"BrownianHydrodynamicsEulerMaruyamaGPU.cuh"
#include<thrust/device_ptr.h>
#include<thrust/device_vector.h>
#include<thrust/for_each.h>
#include<thrust/iterator/zip_iterator.h>




using namespace thrust;

#define TPB 128
namespace brownian_hy_euler_maruyama_ns{
  __constant__ Params params;


  void initGPU(Params m_params){
    m_params.sqrt2dt = sqrt(2.0f)*m_params.sqrtdt;
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(params), &m_params, sizeof(Params)));
  }


  __global__ void float4_to_float3D(const float4 __restrict__ *v4, float3 __restrict__ *v3, uint N){
    uint i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i>=N) return;
    v3[i] = make_float3(v4[i]);
  
  }

  void float4_to_float3GPU(float4 *v4, float3* v3, uint N){
    uint nthreads = TPB<N?TPB:N;
    uint nblocks = N/nthreads +  ((N%nthreads!=0)?1:0); 
    
    float4_to_float3D<<<nthreads,nblocks>>>(v4, v3, N);
  }



  __global__ void integrateGPUD(float4 __restrict__  *pos,
				const float3 __restrict__  *DF,
				const float3 __restrict__ *BdW,
				const float4 __restrict__ *K){
    uint i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i>=params.N) return;
    /*Half step velocity*/
    float4 p = pos[i];
    float4 KR;
    KR.x = params.dt*(dot(K[0], p));
    KR.y = params.dt*(dot(K[1], p));
    KR.z = params.dt*(dot(K[2], p));
    KR.w = 0.0f;
    
    pos[i] += KR + make_float4(DF[i]) + params.sqrt2dt*make_float4(BdW[i]);
    
  }

  //Update the positions
  void integrateGPU(float4 *pos,
		    float3* DF, float3* BdW,
		    float4* K, uint N){
    uint nthreads = TPB<N?TPB:N;
    uint nblocks = N/nthreads +  ((N%nthreads!=0)?1:0); 
    integrateGPUD<<<nblocks, nthreads>>>(pos, DF, BdW, K);    
  }







  __global__ void rotneGPU_prev(float *D, float4 *R, uint N){
    int i_id = blockIdx.x*blockDim.x + threadIdx.x;
    uint n = 3*N;
    for(int id = i_id; id<n*n; id += blockDim.x*gridDim.x){
      /*Compute one pair per thread in the gride-stride loop*/
      /*Get the pair*/
      int j = id/N;
      int i = id%N;
  
      /*Fix the Diagonal boxes of D*/
      float D0 = params.D0;
      if(i >= N || j>=N ||  j<i) continue;
      else if(j==i){
	for(int k = 0; k < 3; k++)
	  for(int l = 0; l < 3; l++){
	    D[3*i + k + n*(3*i + l)] =  0.0f;
	  }
	D[3*i + 0 + n*(3*i + 0)] = D0;
	D[3*i + 1 + n*(3*i + 1)] = D0;
	D[3*i + 2 + n*(3*i + 2)] = D0;
	continue;
      }
      float rh = params.rh;

  
      float4 rij;
      float *rijp = &(rij.x);
    

      float c1, c2;
  
      rij = R[j] - R[i];
      rij.w = 0.0f;
  
      float invr2 = 1.0f/dot(rij, rij);

      float invr = sqrt(invr2);

   
      /*Rotne-Prager-Yamakawa tensor */
      // float r = 1.0f/invr;
      // if(r>=2.0f*rh){
      //   c1 = 0.75f*rh*invr*(1.0f + 2.0f*invr2*rh*rh/3.0f);
      //   c2 = 0.75f*rh*invr*(1.0f - 2.0f*rh*rh*invr2);
      // }
      // else{
      //   c1 = 1.0f - 9.0f*r/(32.0f*rh);
      //   c2 = 3.0f*r/(32.0f*rh);
      // }

      /*Oseen tensor*/
      c1 = 0.75f*invr*rh;
      c2 = c1;

      for(int k = 0; k < 3; k++)
	for(int l = 0; l < 3; l++)
	  D[3*i + k + n*(3*j + l)] = D0*c2*rijp[k]*rijp[l]*invr2;

      for(int k = 0; k<3; k++)  D[3*i + k + n*(3*j + k)] += D0*c1;

    }
  }


  void rodne_callGPU(float *d_D, float4 *d_R, hipStream_t stream, uint N){
    rotneGPU_prev<<<N*N/128/2+1, 128, 0 ,stream>>>(d_D, d_R, N);
  }

















  // __global__ void fix_B(float *B, uint n){
  
  //   uint ii = blockIdx.x*blockDim.x + threadIdx.x;
  //   if(ii >= n*n) return;
  //   uint i = ii%n;
  //   uint j = ii/n;
  //   if(i<=j) return;  
  //   B[ii] = 0.0f;
  // }


  // void fix_B_call(float *B, uint N, hipStream_t stream){
  //   fix_B<<<(3*N*3*N)/128+1,128,0, stream>>>(B, 3*N);
  // }







  // __global__ void rotneGPU(float *D, float3 *R2, uint N){
  //   int id = blockIdx.x*blockDim.x + threadIdx.x;
  
  //   int j = id/N;
  //   int i = id%N;
  //   float D0 = 1.0f;
  //     uint n = 3*N;
  //   if(i >= N || j>=N) return;
  //   else if(j==i){
  //     for(int k = 0; k < 3; k++)
  //       for(int l = 0; l < 3; l++){
  // 	D[3*i + k + n*(3*i + l)] =  k==l?D0:0.0f;
  //       }
  //     return;
  //   }
  //   float rh = 1;
  //   float *R = (float*)R2;

  //   float rij[3];
  //   float r2 = 0.0f;
  //   float r;
  //   float c1, c2;
  //   for(int k = 0; k<3; k++){
  //     rij[k] = R[3*j + k] - R[3*i+k];
  //     r2 += rij[k]*rij[k];
  //   }
  //   r = sqrt(r2);
  //   if(r>=2*rh){
  //     c1 = 0.75*rh/r*(1.0f + 2.0f*rh*rh/(3.0f*r2));
  //     c2 = 0.75*rh/r*(1.0f - 2.0f*rh*rh/r2);
  //   }
  //   else{
  //     c1 = 1.0f - 9.0f*r/(32.0f*rh);
  //     c2 = 3.0f*r/(32.0f*rh);
  //   }

  //   for(int k = 0; k < 3; k++)
  //     for(int l = 0; l < 3; l++)
  //       D[3*i + k + n*(3*j + l)] = D0*c1*(k==l?1.0f:0.0f) + c2*rij[k]*rij[l]/r2;

  // }

}
