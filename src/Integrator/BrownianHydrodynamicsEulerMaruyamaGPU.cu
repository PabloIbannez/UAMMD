#include "hip/hip_runtime.h"

/*Raul P. Pelaez 2016. Brownian Euler Maruyama with hydrodynamics integrator GPU kernels and callers

  Functions to integrate movement. The integration is done via a functor wich accesor ()
     takes a thrust::Tuple containing positions, velocities and forces on each particle. 

TODO:
100-Optimize, see .cpp
*/

#include"utils/hip/hip_vector_types.h"
#include"utils/helper_gpu.cuh"
#include"BrownianHydrodynamicsEulerMaruyamaGPU.cuh"
#include<thrust/device_ptr.h>
#include<thrust/device_vector.h>
#include<thrust/for_each.h>
#include<thrust/iterator/zip_iterator.h>


using namespace thrust;

__constant__ BrownianHydrodynamicsEulerMaruyamaParameters BEMParamsGPU;


void initBrownianHydrodynamicsEulerMaruyamaGPU(BrownianHydrodynamicsEulerMaruyamaParameters m_params){
  m_params.sqrt2dt = sqrt(2.0f)*m_params.sqrtdt;
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(BEMParamsGPU), &m_params, sizeof(BrownianHydrodynamicsEulerMaruyamaParameters)));
}


struct copy_pos_functor{
  copy_pos_functor(){}
  __device__ float3 operator() (const float4& pos4){
    return make_float3(pos4);
  }
};

void copy_pos(float4 *pos, float3* pos3, float4 *force, float3* force3, uint N){
  
  device_ptr<float4> pos4(pos);
  device_ptr<float4> force4(force);
  
  device_ptr<float3> pos3d(pos3);
  device_ptr<float3> force3d(force3);
  
  transform(pos4, pos4+N, pos3d, copy_pos_functor());
  transform(force4, force4+N, force3d, copy_pos_functor());
  
}



//This struct is a thrust trick to perform an arbitrary transformation
//In this case it performs a brownian euler maruyama integration
struct brownianHydroEulerMaruyama_functor{
  float dt;
  __host__ __device__ brownianHydroEulerMaruyama_functor(float dt):
    dt(dt){}
  //The operation is performed on creation
  template <typename Tuple>
  __device__  void operator()(Tuple t){
    /*Retrive the data*/
    float4 pos = get<0>(t);
    float4 DF = make_float4(get<1>(t),0.0f);
    float4 BdW = make_float4(get<2>(t),0.0f);
    float4 KR = make_float4(get<3>(t),0.0f);
    
    pos += KR + DF + BEMParamsGPU.sqrt2dt*BdW;

    get<0>(t) = pos;
  }
};


//Update the positions
void integrateBrownianHydrodynamicsEulerMaruyamaGPU(float4 *pos, float3* DF, float3* BdW, float3* KR,
						    float dt, uint N){
  device_ptr<float4> d_pos4(pos);
  device_ptr<float3> d_DF3(DF);
  device_ptr<float3> d_BdW3(BdW);
  device_ptr<float3> d_KR3(KR);

  /**Thrust black magic to perform a general transformation, see the functor description**/
  for_each(
	   make_zip_iterator( make_tuple( d_pos4, d_DF3, d_BdW3, d_KR3)),
	   make_zip_iterator( make_tuple( d_pos4+N, d_DF3+N, d_BdW3+N, d_KR3+N)),
	   brownianHydroEulerMaruyama_functor(dt));
  //cudaCheckErrors("Integrate");					   
}





__global__ void fix_B(float *B, uint n){
  
  uint ii = blockIdx.x*blockDim.x + threadIdx.x;
  if(ii >= n*n) return;
  uint i = ii%n;
  uint j = ii/n;
  if(i<=j) return;  
  B[ii] = 0.0f;
}


void fix_B_call(float *B, uint N, hipStream_t stream){
  fix_B<<<(3*N*3*N)/128+1,128,0, stream>>>(B, 3*N);
}







__global__ void rotneGPU(float *D, float3 *R2, uint N){
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  
  int j = id/N;
  int i = id%N;
  float D0 = 1.0f;
    uint n = 3*N;
  if(i >= N || j>=N) return;
  else if(j==i){
    for(int k = 0; k < 3; k++)
      for(int l = 0; l < 3; l++){
	D[3*i + k + n*(3*i + l)] =  k==l?D0:0.0f;
      }
    return;
  }
  float rh = 1;
  float *R = (float*)R2;

  float rij[3];
  float r2 = 0.0f;
  float r;
  float c1, c2;
  for(int k = 0; k<3; k++){
    rij[k] = R[3*j + k] - R[3*i+k];
    r2 += rij[k]*rij[k];
  }
  r = sqrt(r2);
  if(r>=2*rh){
    c1 = 0.75*rh/r*(1.0f + 2.0f*rh*rh/(3.0f*r2));
    c2 = 0.75*rh/r*(1.0f - 2.0f*rh*rh/r2);
  }
  else{
    c1 = 1.0f - 9.0f*r/(32.0f*rh);
    c2 = 3.0f*r/(32.0f*rh);
  }

  for(int k = 0; k < 3; k++)
    for(int l = 0; l < 3; l++)
      D[3*i + k + n*(3*j + l)] = D0*c1*(k==l?1.0f:0.0f) + c2*rij[k]*rij[l]/r2;

}















__global__ void rotneGPU_prev(float *D, float3 *R, uint N){
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  //int i = blockIdx.x;
  //int j = threadIdx.x;
  int j = id/N;
  int i = id%N;
  float D0 = 1.0f;
    uint n = 3*N;
  if(i >= N || j>=N ||  j<i) return;
  else if(j==i){
    for(int k = 0; k < 3; k++)
      for(int l = 0; l < 3; l++){
	D[3*i + k + n*(3*i + l)] =  0.0f;
      }
    D[3*i + 0 + n*(3*i + 0)] = D0;
    D[3*i + 1 + n*(3*i + 1)] = D0;
    D[3*i + 2 + n*(3*i + 2)] = D0;
    return;
  }
  float rh = 1;

  
  float3 rij;
  float r;
  float c1, c2;
  
  rij = R[j] - R[i];
  
  float invr2 = 1.0f/dot(rij, rij);

  
  float *rijp = &(rij.x);
  float invr = sqrt(invr2);
  
  r = 1.0f/invr;
  
  if(r>=2*rh){
    c1 = 0.75*rh*invr*(1.0f + 2.0f*invr2*rh*rh/3.0f);
    c2 = 0.75*rh*invr*(1.0f - 2.0f*rh*rh*invr2);
  }
  else{
    c1 = 1.0f - 9.0f*r/(32.0f*rh);
    c2 = 3.0f*r/(32.0f*rh);
  }

  
  for(int k = 0; k < 3; k++)
    for(int l = 0; l < 3; l++)
      D[3*i + k + n*(3*j + l)] = c2*rijp[k]*rijp[l]*invr2;

  for(int k = 0; k<3; k++)  D[3*i + k + n*(3*j + k)] += D0*c1; 
}


void rodne_call(float *d_D, float3 *d_R, hipStream_t stream, uint N){
  rotneGPU_prev<<<N*N/128+1, 128, 0 ,stream>>>(d_D, d_R, N);
}







