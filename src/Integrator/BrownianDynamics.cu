#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2017. Brownian Euler Maruyama Integrator definition

  Solves the following differential equation:
      X[t+dt] = dt(K·X[t]+M·F[t]) + sqrt(2*Tdt)·dW·B
   Being:
     X - Positions
     M - Self Diffusion  coefficient -> 1/(6·pi·vis·radius)
     K - Shear matrix
     dW- Noise vector
     B - sqrt(M)
*/
#include"BrownianDynamics.cuh"

namespace uammd{
  namespace BD{

    EulerMaruyama::EulerMaruyama(shared_ptr<ParticleData> pd,
				 shared_ptr<ParticleGroup> pg,
				 shared_ptr<System> sys,
				 Parameters par):
      Integrator(pd, pg, sys, "[BD::EulerMaruyama]"),
      Kx(make_real3(0)),
      Ky(make_real3(0)),
      Kz(make_real3(0)),
      temperature(par.temperature),
      dt(par.dt),
      is2D(par.is2D),
      steps(0){

      sys->log<System::MESSAGE>("[BD::EulerMaruyama] Initialized");
      
      int numberParticles = pg->getNumberParticles();

      if(temperature>real(0.0)){
	noise.resize(3*numberParticles + ((3*numberParticles)%2));
	hiprandCreateGenerator(&curng, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(curng, sys->rng().next());
	auto d_noise = thrust::raw_pointer_cast(noise.data());
	hiprandGenerateNormal(curng, d_noise, noise.size(),  0.0, 1.0);	
      }


      this->selfDiffusion = 1.0/(6.0*M_PI*par.viscosity);

      sys->log<System::MESSAGE>("[BD::EulerMaruyama] Temperature: %f", temperature);
      sys->log<System::MESSAGE>("[BD::EulerMaruyama] dt: %f", dt);

      
      if(par.hydrodynamicRadius != real(-1.0)){
	this->selfDiffusion /= par.hydrodynamicRadius;
	this->hydrodynamicRadius = par.hydrodynamicRadius;
	if(pd->isRadiusAllocated()){
	  sys->log<System::WARNING>("[BD::EulerMaruyama] Assuming all particles have hydrodynamic radius %f",
				    par.hydrodynamicRadius);
	}
	else{
	  sys->log<System::MESSAGE>("[BD::EulerMaruyama] Hydrodynamic radius: %f", par.hydrodynamicRadius);
	}
	sys->log<System::MESSAGE>("[BD::EulerMaruyama] Self Diffusion: %f", selfDiffusion);
      }
      else if(pd->isRadiusAllocated()){
	sys->log<System::MESSAGE>("[BD::EulerMaruyama] Hydrodynamic radius: particleRadius");
	sys->log<System::MESSAGE>("[BD::EulerMaruyama] Self Diffusion: %f/particleRadius",
				    selfDiffusion);      
      }
      else{
	//Default hydrodynamic radius when none is provided is 1
	this->hydrodynamicRadius = real(1.0);
	sys->log<System::MESSAGE>("[BD::EulerMaruyama] Hydrodynamic radius: %f", hydrodynamicRadius);
	sys->log<System::MESSAGE>("[BD::EulerMaruyama] Self Diffusion: %f", selfDiffusion);
      }      


      this->sqrt2MTdt = sqrt(2.0*selfDiffusion*temperature*dt);



      if(par.K.size()==3){
	Kx = par.K[0];
	Ky = par.K[1];
	Kz = par.K[2];
	sys->log<System::MESSAGE>("[BD::EulerMaruyama] Shear Matrix: [ %f %f %f; %f %f %f; %f %f %f ]",
				  Kx.x, Kx.y, Kx.z,
				  Ky.x, Ky.y, Ky.z,
				  Kz.x, Kz.y, Kz.z);
      }
      
      if(is2D){
	sys->log<System::MESSAGE>("[BD::EulerMaruyama] Starting in 2D mode");
      }

      hipStreamCreate(&noiseStream);
      hipStreamCreate(&forceStream);

    }
    

    EulerMaruyama::~EulerMaruyama(){
      sys->log<System::MESSAGE>("[BD::EulerMaruyama] Destroyed");
      if(this->is2D)hiprandDestroyGenerator(curng);
      hipStreamDestroy(noiseStream);
      hipStreamDestroy(forceStream);		     
    }

    
    namespace EulerMaruyama_ns{
      /*Integrate the movement*/
      __global__ void integrateGPU(real4 __restrict__  *pos,
				   ParticleGroup::IndexIterator indexIterator,
				   const real4 __restrict__  *force,
				   const real3 __restrict__ *dW,
				   real3 Kx, real3 Ky, real3 Kz,
				   real selfDiffusion,
				   real * radius,
				   real dt,
				   bool is2D,
				   real sqrt2MTdt,
				   int N){
	uint id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id>=N) return;

	int i = indexIterator[id];
	/*Half step velocity*/	
	real3 p = make_real3(pos[i]);
	real3 f = make_real3(force[i]);

	real3 KR = make_real3(dot(Kx, p),
			      dot(Ky, p),
			      dot(Kz, p));

	real invRadius = real(1.0);
	if(radius) invRadius = real(1.0)/radius[i];
	// X[t+dt] = dt(K·X[t]+M·F[t]) + sqrt(2·T·dt)·dW·B
	p += dt*( KR + selfDiffusion*invRadius*f);
	if(dW){ //When temperature > 0
	  real sqrtInvRadius = real(1.0);
	  if(radius) sqrtInvRadius = sqrtf(invRadius);
	  p += sqrt2MTdt*dW[i]*sqrtInvRadius;
	}

	pos[i].x = p.x;
	pos[i].y = p.y;
	if(!is2D)
	  pos[i].z = p.z;
      }

    }

    void EulerMaruyama::forwardTime(){
      steps++;
      sys->log<System::DEBUG1>("[BD::EulerMaruyama] Performing integration step %d", steps);

      for(auto forceComp: interactors) forceComp->updateSimulationTime(steps*dt);

      if(steps==1){
	for(auto forceComp: interactors){
	  forceComp->updateTimeStep(dt);
	  forceComp->updateTemperature(temperature);	 
	}
      }
      int numberParticles = pg->getNumberParticles();
      int BLOCKSIZE = 128;
      uint Nthreads = BLOCKSIZE<numberParticles?BLOCKSIZE:numberParticles;
      uint Nblocks = numberParticles/Nthreads +  ((numberParticles%Nthreads!=0)?1:0);

      real * d_radius = nullptr;
      if(hydrodynamicRadius == real(-1.0) && pd->isRadiusAllocated()){
	auto radius = pd->getRadius(access::location::gpu, access::mode::read);
	d_radius = radius.raw();
      }
      

      real3 * d_noise = nullptr;
      if(temperature > real(0.0)){
	hiprandSetStream(curng, noiseStream);
	noise.resize(3*numberParticles + ((3*numberParticles)%2));
	d_noise = (real3*)thrust::raw_pointer_cast(noise.data());
	hiprandGenerateNormal(curng, (real*)d_noise, noise.size(),  0.0, 1.0);	
      }

      
      /*Compute new forces*/
      auto groupIterator = pg->getIndexIterator(access::location::gpu);
      {
	auto force = pd->getForce(access::location::gpu, access::mode::write);      
	fillWithGPU<<<Nblocks, Nthreads>>>(force.raw(), groupIterator, make_real4(0), numberParticles);
      }
      for(auto forceComp: interactors) forceComp->sumForce(forceStream);
    
      

      auto pos = pd->getPos(access::location::gpu, access::mode::readwrite);
      auto force = pd->getForce(access::location::gpu, access::mode::read);
      EulerMaruyama_ns::integrateGPU<<<Nblocks, Nthreads>>>(pos.raw(),
							    groupIterator,
							    force.raw(),
							    d_noise,
							    Kx, Ky, Kz,
							    selfDiffusion,
							    d_radius,
							    dt,
							    is2D,
							    sqrt2MTdt,
							    numberParticles);

    }
    

  }
}