#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2018. Force Coupling Method BDHI Module.

  This code implements the algorithm described in [1], using cuFFT to solve te velocity in eq. 24 of [1] and compute the brownian fluctuations of eq. 30 in [1] (it only needs two FFT's). It only includes the stokeslet terms.

  This code is adapted from PSE, basically the factor sinc(ka/2)^2 is removed from the kernel and the near part is removed. Also the spreading/interpolation kernel is now an exponential with different support and std.

  The operator terminology used in the comments (as well as the wave space part of the algorithm) comes from [2], the PSE basic reference.
References:
[1] Fluctuating force-coupling method for simulations of colloidal suspensions. Eric E. Keaveny. 2014.
[2]  Rapid Sampling of Stochastic Displacements in Brownian Dynamics Simulations. Fiore, Balboa, Donev and Swan. 2017.

*/
#include"BDHI_FCM.cuh"
#include"utils/GPUUtils.cuh"
#include"third_party/saruprng.cuh"
#include<vector>
#include<algorithm>
#include<fstream>
#include"utils/debugTools.cuh"
#include"utils/cufftDebug.h"
#include"utils/cxx_utils.h"

namespace uammd{
  namespace BDHI{

    namespace FCM_ns{

      //Looks for the closest (equal or greater) number of nodes of the form 2^a*3^b*5^c
      int3 nextFFTWiseSize3D(int3 size){
	
	int* cdim = &size.x;

	int max_dim = std::max({size.x, size.y, size.z});
	
	int n= 5;
	std::vector<int> tmp(n*n*n*n*n, 0);
	do{
	  tmp.resize(n*n*n*n*n, 0);
	  fori(0,n)forj(0,n)for(int k=0; k<n;k++)for(int k7=0; k7<n; k7++)for(int k11=0; k11<n; k11++){
		int id = i+n*j+n*n*k+n*n*n*k7+n*n*n*n*k11;
		tmp[id] = 0;
		//Current fft wise size
		int number = pow(2,i)*pow(3,j)*pow(5,k)*pow(7, k7)*pow(11, k11);
		//The fastest FFTs always have at least a factor of 2
		if(i==0) continue;
		//I have seen empiracally that factor 11 and 7 only works well with at least a factor 2 involved
		if((k11>0 && (i==0))) continue;
		tmp[id] = number;
	      }
	  n++;
	  /*Sort this array in ascending order*/
	  std::sort(tmp.begin(), tmp.end());      
	}while(tmp.back()<max_dim); /*if n=5 is not enough, include more*/
	
	//I have empirically seen that these sizes produce slower FFTs than they should in several platforms
	constexpr int forbiddenSizes [] = {28, 98, 150, 154, 162, 196, 242};
	/*Now look for the nearest value in tmp that is greater than each cell dimension and it is not forbidden*/
	forj(0,3){
	  fori(0, tmp.size()){	    
	    if(tmp[i]<cdim[j]) continue;
	    for(int k =0;k<sizeof(forbiddenSizes)/sizeof(int); k++) if(tmp[i] == forbiddenSizes[k]) continue;
	    cdim[j] = tmp[i];
	    break;
	  }
	    

	  
	}
	return size;
      }
      /*This function takes a node index and returns the corresponding wave number*/
      template<class vec3>
      inline __device__ vec3 cellToWaveNumber(const int3 &cell, const int3 &cellDim, const vec3 &L){
	const vec3 pi2invL = real(2.0)*real(M_PI)/L;
	/*My wave number*/
	vec3 k = {cell.x*pi2invL.x,
		  cell.y*pi2invL.y,
		  cell.z*pi2invL.z};
	/*Be careful with the conjugates*/
	/*Remember that FFT stores wave numbers as K=0:N/2+1:-N/2:-1 */
	if(cell.x >= (cellDim.x+1)/2) k.x -= real(cellDim.x)*pi2invL.x;
	if(cell.y >= (cellDim.y+1)/2) k.y -= real(cellDim.y)*pi2invL.y;
	if(cell.z >= (cellDim.z+1)/2) k.z -= real(cellDim.z)*pi2invL.z;
	return k;
      }

    }
    /*Constructor*/
    FCM::FCM(shared_ptr<ParticleData> pd,
	     shared_ptr<ParticleGroup> pg,
	     shared_ptr<System> sys,
	     Parameters par):
      pd(pd), pg(pg), sys(sys),
      dt(par.dt),
      temperature(par.temperature),
      viscosity(par.viscosity),
      box(par.box), grid(box, int3()){
      
      if(box.boxSize.x == real(0.0) && box.boxSize.y == real(0.0) && box.boxSize.z == real(0.0)){
	sys->log<System::CRITICAL>("[BDHI::FCM] Box of size zero detected, cannot work without a box! (make sure a box parameter was passed)");
      }
      if(box.boxSize.x != box.boxSize.y || box.boxSize.y != box.boxSize.z || box.boxSize.x != box.boxSize.z){
	sys->log<System::WARNING>("[BDHI::FCM] Non cubic boxes are not really tested!");
      }


      seed = sys->rng().next();
      sys->log<System::MESSAGE>("[BDHI::FCM] Initialized");

      int numberParticles = pg->getNumberParticles();

      sys->log<System::MESSAGE>("[BDHI::FCM] Self mobility: %f", 1.0/(6*M_PI*par.viscosity*par.hydrodynamicRadius)*(1-2.837297*par.hydrodynamicRadius/box.boxSize.x));
  
      
      this->sigma = par.hydrodynamicRadius/sqrt(M_PI); //eq. 8 in [1], \sigma_\Delta
      int3 cellDim;
      if(par.cells.x<=0){
	double minFactor = 1.86; //According to [1] \sigma_\Delta/H = 1.86 gives enough accuracy
	real h = sigma/minFactor;
	cellDim = FCM_ns::nextFFTWiseSize3D(make_int3(box.boxSize/h));
      }
      else{      
        cellDim = par.cells;
      }
      grid = Grid(box, cellDim);
      
      //According to [1] the Gaussian kernel can be considered 0 beyond 3*a, so P >= 3*a/h
      this->P = make_int3(3*par.hydrodynamicRadius/grid.cellSize.x+0.5); 
      if(this->P.x < 1 ) this->P = make_int3(1);

      
      sys->log<System::MESSAGE>("[BDHI::FCM] Box Size: %f %f %f", box.boxSize.x, box.boxSize.y, box.boxSize.z);
      sys->log<System::MESSAGE>("[BDHI::FCM] Far range grid size: %d %d %d", cellDim.x, cellDim.y, cellDim.z);
      sys->log<System::MESSAGE>("[BDHI::FCM] Gaussian kernel support: %d", 2*this->P.x+1);
      sys->log<System::MESSAGE>("[BDHI::FCM] σ_Δ: %f", sigma);
      sys->log<System::MESSAGE>("[BDHI::FCM] h: %f", grid.cellSize.x);
      sys->log<System::MESSAGE>("[BDHI::FCM] σ_Δ/h: %f", sigma/grid.cellSize.x);
      sys->log<System::MESSAGE>("[BDHI::FCM] Cell volume: %e", grid.cellSize.x*grid.cellSize.y*grid.cellSize.z);
      
      CudaSafeCall(hipStreamCreate(&stream));
      CudaSafeCall(hipStreamCreate(&stream2));
  
      /*The quantity spreaded to the grid in real or wave space*/
      /*The layout of this array is
	fx000, fy000, fz000, fx001, fy001, fz001..., fxnnn, fynnn, fznnn. n=ncells-1
	When used in real space each f is a real number, whereas in wave space each f will be a complex number.
	See cufftC2R of R2C in place in Mdot_far
      */
      /*Can be Force when spreading particles to the grid and
	velocities when interpolating from the grid to the particles*/
      int ncells = grid.cellDim.x*grid.cellDim.y*grid.cellDim.z;
      gridVelsFourier.resize(3*ncells, hipfftComplex());
            
      initCuFFT();
  
      CudaSafeCall(hipDeviceSynchronize());
      CudaCheckError();
    }

    void FCM::initCuFFT(){
      CufftSafeCall(hipfftCreate(&cufft_plan_forward));
      CufftSafeCall(hipfftCreate(&cufft_plan_inverse));
      
      /*I will be handling workspace memory*/
      CufftSafeCall(hipfftSetAutoAllocation(cufft_plan_forward, 0));
      CufftSafeCall(hipfftSetAutoAllocation(cufft_plan_inverse, 0));

      //Required storage for the plans
      size_t cufftWorkSizef = 0, cufftWorkSizei = 0;
      /*Set up cuFFT*/
      int3 cdtmp = {grid.cellDim.x, grid.cellDim.y, grid.cellDim.z};
      /*I want to make three 3D FFTs, each one using one of the three interleaved coordinates*/
      CufftSafeCall(hipfftMakePlanMany(cufft_plan_forward,
				      3, &cdtmp.x, /*Three dimensional FFT*/
				      &cdtmp.x,
				      /*Each FFT starts in 1+previous FFT index. FFTx in 0*/
				      3, 1, //Each element separated by three others x0 y0 z0 x1 y1 z1...
				      /*Same format in the output*/
				      &cdtmp.x,
				      3, 1,
				      /*Perform 3 direct Batched FFTs*/
				      HIPFFT_R2C, 3,
				      &cufftWorkSizef));

      sys->log<System::DEBUG>("[BDHI::FCM] cuFFT grid size: %d %d %d", cdtmp.x, cdtmp.y, cdtmp.z);
      /*Same as above, but with C2R for inverse FFT*/
      CufftSafeCall(hipfftMakePlanMany(cufft_plan_inverse,
				      3, &cdtmp.x, /*Three dimensional FFT*/
				      &cdtmp.x,
				      /*Each FFT starts in 1+previous FFT index. FFTx in 0*/
				      3, 1, //Each element separated by three others x0 y0 z0 x1 y1 z1...
				      &cdtmp.x,
				      3, 1,
				      /*Perform 3 inverse batched FFTs*/
				      HIPFFT_C2R, 3,
				      &cufftWorkSizei));

      /*Allocate cuFFT work area*/
      size_t cufftWorkSize = std::max(cufftWorkSizef, cufftWorkSizei);

      sys->log<System::DEBUG>("[BDHI::FCM] Necessary work space for cuFFT: %s", printUtils::prettySize(cufftWorkSize).c_str());
      size_t free_mem, total_mem;
      CudaSafeCall(hipMemGetInfo(&free_mem, &total_mem));

      if(free_mem<cufftWorkSize){
	sys->log<System::CRITICAL>("[BDHI::FCM] Not enough memory in device to allocate cuFFT free %s, needed: %s!!",
				   printUtils::prettySize(free_mem).c_str(),
				   printUtils::prettySize(cufftWorkSize).c_str());
      }

      cufftWorkArea.resize(cufftWorkSize/sizeof(real)+1);
      auto d_cufftWorkArea = thrust::raw_pointer_cast(cufftWorkArea.data());
      
      CufftSafeCall(hipfftSetWorkArea(cufft_plan_forward, (void*)d_cufftWorkArea));
      CufftSafeCall(hipfftSetWorkArea(cufft_plan_inverse, (void*)d_cufftWorkArea));
    }

    FCM::~FCM(){
      CudaSafeCall(hipDeviceSynchronize());
      CufftSafeCall(hipfftDestroy(cufft_plan_inverse));
      CufftSafeCall(hipfftDestroy(cufft_plan_forward));
      CudaSafeCall(hipStreamDestroy(stream));
      CudaSafeCall(hipStreamDestroy(stream2));
    }

    //I dont need to do anything at the begining of a step
    void FCM::setup_step(hipStream_t st){}

    //Compute M·v = Mw·v
    template<typename vtype>
    void FCM::Mdot(real3 *Mv, vtype *v, hipStream_t st){
      sys->log<System::DEBUG1>("[BDHI::FCM] Mdot....");
      {
	int numberParticles = pg->getNumberParticles();
	int BLOCKSIZE = 128;
	int Nthreads = BLOCKSIZE<numberParticles?BLOCKSIZE:numberParticles;
	int Nblocks  =  numberParticles/Nthreads +  ((numberParticles%Nthreads!=0)?1:0); 

	fillWithGPU<<<Nblocks, Nthreads>>>(Mv, make_real3(0.0), numberParticles);
      }
      Mdot_far<vtype>(Mv, v, st);
      
    }
    namespace FCM_ns{
      using cufftComplex3 = FCM::cufftComplex3;
#ifndef SINGLE_PRECISION
      __device__ double atomicAdd(double* address, double val){
	unsigned long long int* address_as_ull =
	  (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
	  assumed = old;
	  old = atomicCAS(address_as_ull, assumed,
			  __double_as_longlong(val +
					       __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
      }
#endif      
      /*Apply the projection operator to a wave number with a certain complex factor.
	res = (I-\hat{k}^\hat{k})·factor*/

      inline __device__ cufftComplex3 projectFourier(const real3 &k, const cufftComplex3 &factor){
	const real invk2 = real(1.0)/dot(k,k);

	cufftComplex3 res;
	{//Real part
	  const real3 fr = make_real3(factor.x.x, factor.y.x, factor.z.x);
	  const real kfr = dot(k,fr)*invk2;
	  const real3 vr = (fr-k*kfr);
	  res.x.x = vr.x;
	  res.y.x = vr.y;
	  res.z.x = vr.z;
	}
	{//Imaginary part
	  const real3 fi = make_real3(factor.x.y, factor.y.y, factor.z.y);
	  const real kfi = dot(k,fi)*invk2;
	  const real3 vi = (fi-k*kfi);
	  res.x.y = vi.x;
	  res.y.y = vi.y;
	  res.z.y = vi.z;	  
	}
	return res;
      }
      
      /*Spreads the 3D quantity v (i.e the force) to a regular grid
	For that it uses a Gaussian kernel of the form f(r) = prefactor·exp(-tau·r^2). See eq. 8 in [1]
	i.e. Applies the operator S.
	Launch a block per particle.
      */
      template<typename vtype> /*Can take a real3 or a real4*/
      __global__ void particles2GridD(real4 * __restrict__ pos, /*Particle positions*/
				      vtype * __restrict__ v,   /*Per particle quantity to spread*/
				      real3 * __restrict__ gridVels, /*Interpolated values, size ncells*/
				      int N, /*Number of particles*/
				      int3 P, /*Gaussian kernel support in each dimension*/
				      Grid grid, /*Grid information and methods*/
				      real3 prefactor,/*Prefactor for the kernel*/
				      real3 tau /*Kernel exponential factor*/){
	const int id = blockIdx.x;
	const int tid = threadIdx.x;
	if(id>=N) return;

	/*Get pos and v (i.e force)*/
	__shared__ real3 pi;
	__shared__ real3 vi_pf;
	__shared__ int3 celli;
	if(tid==0){
	  pi = make_real3(pos[id]);
	  vi_pf = make_real3(v[id])*prefactor;
	  /*Get my cell*/
	  celli = grid.getCell(pi);	  
	}
	/*Conversion between cell number and cell center position*/
	const real3 cellPosOffset = real(0.5)*(grid.cellSize - grid.box.boxSize);
	const int3 supportCells = 2*P + 1;
	const int numberNeighbourCells = supportCells.x*supportCells.y*supportCells.z;

	__syncthreads();
	for(int i = tid; i<numberNeighbourCells; i+=blockDim.x){
	  /*Compute neighbouring cell*/
	  int3 cellj = make_int3(celli.x + i%supportCells.x - P.x,
				 celli.y + (i/supportCells.x)%supportCells.y - P.y,
				 celli.z + i/(supportCells.x*supportCells.y) - P.z );
	  cellj = grid.pbc_cell(cellj);
	  
	  /*Distance from particle i to center of cell j*/
	  const real3 rij = grid.box.apply_pbc(pi-make_real3(cellj)*grid.cellSize-cellPosOffset);	  
	  const real r2 = dot(rij, rij);

	  /*The weight of particle i on cell j*/
	  const real3 weight = vi_pf*make_real3(exp(-r2*tau.x), exp(-r2*tau.y), exp(-r2*tau.z));

	  /*Get index of cell j*/
	  const int jcell = grid.getCellIndex(cellj);
	  
	  /*Atomically sum my contribution to cell j*/
	  atomicAdd(&gridVels[jcell].x, weight.x);
	  atomicAdd(&gridVels[jcell].y, weight.y);
	  atomicAdd(&gridVels[jcell].z, weight.z);
	  
	}
      }

      
      /*Scales fourier transformed forces in the regular grid to obtain velocities,
	(Mw·F)_deterministic = σ·St·FFTi·B·FFTf·S·F	
	 Input: gridForces = FFTf·S·F
	 Output:gridVels = B·FFTf·S·F -> B \propto (I-k^k/|k|^2) 
       */
      /*A thread per fourier node*/
      __global__ void forceFourier2Vel(cufftComplex3 * gridForces, /*Input array*/
				       cufftComplex3 * gridVels, /*Output array, can be the same as input*/
				       real vis,
				       real dV,				      
				       Grid grid/*Grid information and methods*/
				       ){
	/*Get my cell*/
	int3 cell;
	cell.x= blockIdx.x*blockDim.x + threadIdx.x;
	cell.y= blockIdx.y*blockDim.y + threadIdx.y;
	cell.z= blockIdx.z*blockDim.z + threadIdx.z;
	/*Only the first half of the innermost dimension is stored, the rest is redundant*/
	if(cell.x>=grid.cellDim.x/2+1) return;
	if(cell.y>=grid.cellDim.y) return;
	if(cell.z>=grid.cellDim.z) return;
	
	const int icell = grid.getCellIndex(cell);
	if(icell == 0){
	  gridVels[0] = {0,0, 0,0, 0,0};
	  return;
	}
	const int ncells = grid.getNumberCells();
	const real3 k = cellToWaveNumber(cell, grid.cellDim, grid.box.boxSize);
	const real invk2 = real(1.0)/dot(k,k);
	/*Get my scaling factor B*/
	const real3 B = make_real3(invk2/(vis*dV*double(ncells)));
	cufftComplex3 factor = gridForces[icell];

	factor.x *= B.x;
	factor.y *= B.y;
	factor.z *= B.z;	
	
	/*Store vel in global memory, note that this is overwritting any previous value in gridVels*/
	gridVels[icell] = projectFourier(k, factor);	  
      }

      /*Computes the long range stochastic velocity term
	Mw·F + sqrt(Mw)·dWw = σ·St·FFTi·B·FFTf·S·F+ √σ·St·FFTi·√B·dWw = 
	= σ·St·FFTi( B·FFTf·S·F + 1/√σ·√B·dWw)
	This kernel gets v_k = gridVelsFourier = B·FFtt·S·F as input and adds 1/√σ·√B(k)·dWw.
	Keeping special care that v_k = v*_{N-k}, which implies that dWw_k = dWw*_{N-k}
	See eq. 30 in [1].
	Launch a thread per cell grid/fourier node
      */
      __global__ void fourierBrownianNoise(/*Values of vels on each cell*/
					   cufftComplex3 *__restrict__ gridVelsFourier, 
					   Grid grid, /*Grid parameters. Size of a cell, number of cells...*/
					   real prefactor,/* sqrt(2·T/dt)*/
					   real vis,
					   //Parameters to seed the RNG					   
					   ullint seed,
					   ullint step
					   ){
	/*Get my cell*/
	int3 cell;
	cell.x= blockIdx.x*blockDim.x + threadIdx.x;
	cell.y= blockIdx.y*blockDim.y + threadIdx.y;
	cell.z= blockIdx.z*blockDim.z + threadIdx.z;
	/*This indesx is computed here to use it as a seed for the RNG*/
	int icell = grid.getCellIndex(cell);
	/*cuFFT R2C and C2R only store half of the innermost dimension, the one that varies the fastest
      
	  The input of R2C is real and the output of C2R is real. 
	  The only way for this to be true is if v_k={i,j,k} = v*_k{N-i, N-j, N-k}

	  So the conjugates are redundant and the is no need to compute them nor store them except on two exceptions.
	  In this scheme, the only cases in which v_k and v_{N-k} are stored are:
	     1- When the innermost dimension coordinate is 0.
	     2- When the innermost dimension coordinate is N/2 and N is even.
	*/
	/*Only compute the first half of the innermost dimension*/
	if(2*cell.x >= grid.cellDim.x+1) return;
	if(cell.y >= grid.cellDim.y) return;
	if(cell.z >= grid.cellDim.z) return;

	const int ncells = grid.getNumberCells();
	/*K=0 is not added, no stochastic motion is added to the center of mass*/	
	if((cell.x == 0 and cell.y == 0 and cell.z == 0) or
	   /*These terms will be computed along its conjugates*/
	   /*These are special because the conjugate of k_i=0 is k_i=N_i, 
	     which is not stored and therfore must not be computed*/
	   (cell.x==0 and cell.y == 0 and 2*cell.z >= grid.cellDim.z+1) or
	   (cell.x==0 and 2*cell.y >= grid.cellDim.y+1)) return;
	    
	/*Compute gaussian complex noise dW, 
	  std = prefactor -> ||z||^2 = <x^2>/sqrt(2)+<y^2>/sqrt(2) = prefactor*/
	/*A complex random number for each direction*/
	cufftComplex3 noise;
	{
	  //Uncomment to use uniform numbers instead of gaussian
	  Saru saru(icell, step, seed);
	  const real complex_gaussian_sc = real(0.707106781186547)*prefactor; //1/sqrt(2)
	  //const real sqrt32 = real(1.22474487139159)*prefactor;
	  // = make_real2(saru.f(-1.0f, 1.0f),saru.f(-1.0f, 1.0f))*sqrt32;
	  noise.x = make_real2(saru.gf(0, complex_gaussian_sc));
	  // = make_real2(saru.f(-1.0f, 1.0f),saru.f(-1.0f, 1.0f))*sqrt32;
	  noise.y = make_real2(saru.gf(0, complex_gaussian_sc));
	  // = make_real2(saru.f(-1.0f, 1.0f),saru.f(-1.0f, 1.0f))*sqrt32;
	  noise.z = make_real2(saru.gf(0, complex_gaussian_sc));
	}
	/*Beware of nyquist points! They only appear with even cell dimensions
	  There are 8 nyquist points at most (cell=0,0,0 is excluded at the start of the kernel)
	  These are the 8 vertex of the inferior left cuadrant. The O points:
               +--------+--------+
              /|       /|       /|
             / |      / |      / | 
            +--------+--------+  |
           /|  |    /|  |    /|  |
          / |  +---/-|--+---/-|--+
         +--------+--------+  |	/|
         |  |/ |  |  |/ |  |  |/ |
         |  O-----|--O-----|--+	 |
         | /|6 |  | /|7 |  | /|	 |
         |/ |  +--|/-|--+--|/-|--+
         O--------O--------+  |	/ 
         |5 |/    |4 |/    |  |/  
         |  O-----|--O-----|--+	  
     ^   | / 3    | / 2    | /  ^ 
     |   |/       |/       |/  /  
     kz  O--------O--------+  ky  
         kx ->     1
	*/
	/*Handle nyquist points*/

	bool nyquist;
	{ //Is the current wave number a nyquist point?
	  bool isXnyquist = (cell.x == grid.cellDim.x - cell.x) && (grid.cellDim.x%2 == 0);
	  bool isYnyquist = (cell.y == grid.cellDim.y - cell.y) && (grid.cellDim.y%2 == 0);
	  bool isZnyquist = (cell.z == grid.cellDim.z - cell.z) && (grid.cellDim.z%2 == 0);

	  nyquist =  (isXnyquist && cell.y==0   && cell.z==0)  or  //1
               	     (isXnyquist && isYnyquist  && cell.z==0)  or  //2
               	     (cell.x==0  && isYnyquist  && cell.z==0)  or  //3
               	     (isXnyquist && cell.y==0   && isZnyquist) or  //4
               	     (cell.x==0  && cell.y==0   && isZnyquist) or  //5
               	     (cell.x==0  && isYnyquist  && isZnyquist) or  //6
               	     (isXnyquist && isYnyquist  && isZnyquist);    //7
	}
	
	if(nyquist){
	  /*Nyquist points are their own conjugates, so they must be real.
	    ||r||^2 = <x^2> = ||Real{z}||^2 = <Real{z}^2>·sqrt(2) =  prefactor*/
	  constexpr real nqsc = real(1.41421356237310); //sqrt(2)
	  noise.x.x *= nqsc; noise.x.y = 0;
	  noise.y.x *= nqsc; noise.y.y = 0;
	  noise.z.x *= nqsc; noise.z.y = 0;
	}
	/*Z = sqrt(B)·(I-k^k)·dW*/
	{// Compute for v_k wave number
	  const real3 k = cellToWaveNumber(cell, grid.cellDim, grid.box.boxSize);

	  const real invk2 = real(1.0)/dot(k,k);
	  /*Get my scaling factor B*/
	  const real3 B = make_real3(invk2/(vis*real(ncells)));
	  const real3 Bsq = sqrt(B);
	  
	  cufftComplex3 factor = noise;
	  factor.x *= Bsq.x;
	  factor.y *= Bsq.y;
	  factor.z *= Bsq.z;	  
	  
	  gridVelsFourier[icell] += projectFourier(k, factor);
	}
	/*Compute for conjugate v_{N-k} if needed*/
	
	/*Take care of conjugate wave number -> v_{Nx-kx,Ny-ky, Nz-kz}*/
	/*The special cases k_i=0 do not have conjugates, a.i N-k = N which is not stored*/
	
	if(nyquist) return; //Nyquist points do not have conjugates

	/*Conjugates are stored only when kx == Nx/2 or kx=0*/	
	if(cell.x == grid.cellDim.x-cell.x or cell.x == 0){
	  /*The only case with x conjugates is when kx = Nx-kx or kx=0, so this line is not needed*/
	  //if(cell.x > 0) cell.x = grid.cellDim.x-cell.x;
	  /*k_i=N_i is not stored, so do not conjugate them, the necessary exclusions are at the start of the kernel*/
	  if(cell.y > 0) cell.y = grid.cellDim.y-cell.y;
	  if(cell.z > 0) cell.z = grid.cellDim.z-cell.z;
	  
	  icell = grid.getCellIndex(cell);
	  
	  const real3 k = cellToWaveNumber(cell, grid.cellDim, grid.box.boxSize);

	  const real invk2 = real(1.0)/dot(k,k);
	  /*Get my scaling factor B*/
	  const real3 B = make_real3(invk2/(vis*real(ncells)));

	  const real3 Bsq = sqrt(B);	  
	  cufftComplex3 factor = noise;
	  /*v_{N-k} = v*_k, so the complex noise must be conjugated*/
	  factor.x.y *= real(-1.0);
	  factor.y.y *= real(-1.0);
	  factor.z.y *= real(-1.0);
	  
	  factor.x *= Bsq.x;
	  factor.y *= Bsq.y;
	  factor.z *= Bsq.z;
	  
	  gridVelsFourier[icell] += projectFourier(k, factor);
	}
      }
      
      /*Interpolates a quantity (i.e velocity) from its values in the grid to the particles.
	For that it uses a Gaussian kernel of the form f(r) = prefactor·exp(-tau·r^2)
	σ = dx*dy*dz; h^3 in [1]
	Mw·F + sqrt(Mw)·dWw = σ·St·FFTi·B·FFTf·S·F+ √σ·St·FFTi·√B·dWw = 
	= σ·St·FFTi( B·FFTf·S·F + 1/√σ·√B·dWw)

	Input: gridVels = FFTi( B·FFTf·S·F + 1/√σ·√B·dWw)
	Output: Mv = σ·St·gridVels
	The first term is computed in forceFourier2Vel and the second in fourierBrownianNoise
      */
      template<typename vtype>
      __global__ void grid2ParticlesD(real4 * __restrict__ pos,
				      vtype * __restrict__ Mv, /*Result (i.e Mw·F)*/
				      real3 * __restrict__ gridVels, /*Values in the grid*/
				      int N, /*Number of particles*/
				      int3 P, /*Gaussian kernel support in each dimension*/
				      Grid grid, /*Grid information and methods*/				  
				      real3 prefactor,/*Prefactor for the kernel, (2*xi*xi/(pi·eta))^3/2*/
				      real3 tau /*Kernel exponential factor, 2*xi*xi/eta*/
				      ){
	/*A thread per particle */
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id>=N) return;
	/*Get my particle and my cell*/
    
	const real3 pi = make_real3(pos[id]);
	const int3 celli = grid.getCell(pi);
	int3 cellj;
	/*The S^T = St = σ S*/    
	prefactor *= (grid.cellSize.x*grid.cellSize.y*grid.cellSize.z);

	real3  result = make_real3(0);
    
	int x,y,z;
	/*Transform cell number to cell center position*/
	real3 cellPosOffset = real(0.5)*(grid.cellSize-grid.box.boxSize);
	/*Transvers the Pth neighbour cells*/
	for(z=-P.z; z<=P.z; z++){
	  cellj.z = grid.pbc_cell_coord<2>(celli.z + z);
	  for(y=-P.y; y<=P.y; y++){
	    cellj.y = grid.pbc_cell_coord<1>(celli.y + y);
	    for(x=-P.x; x<=P.x; x++){
	      cellj.x = grid.pbc_cell_coord<0>(celli.x + x);
	      /*Get neighbour cell*/	  
	      int jcell = grid.getCellIndex(cellj);

	      /*Compute distance to center*/
	      real3 rij = grid.box.apply_pbc(pi-make_real3(cellj)*grid.cellSize - cellPosOffset);
	      real r2 = dot(rij, rij);
	      /*Interpolate cell value and sum*/
	      real3 cellj_vel = make_real3(gridVels[jcell]);
	      result += prefactor*make_real3(exp(-tau.x*r2), exp(-tau.y*r2), exp(-tau.z*r2))*cellj_vel;
	    }
	  }
	}
	/*Write total to global memory*/
	Mv[id] += result;
      }

    }
    
    /*Compute M·F and B·dW in Fourier space
      σ = dx*dy*dz; h^3 in [1]
      Mw·F + sqrt(Mw)·dWw = σ·St·FFTi·B·FFTf·S·F+ √σ·St·FFTi·√B·dWw = 
      = σ·St·FFTi( B·FFTf·S·F + 1/√σ·√B·dWw)
    */
    template<typename vtype>
    void FCM::spreadParticles(vtype *v, hipStream_t st){
      int numberParticles = pg->getNumberParticles();
      auto pos = pd->getPos(access::location::gpu, access::mode::read);
      real3* d_gridVels = (real3*)thrust::raw_pointer_cast(gridVelsFourier.data());
      //Gaussian spreading/interpolation kernel parameters, s(r) = prefactor*exp(-tau*r2)           
      real3 prefactorGaussian = make_real3(pow(2*M_PI*sigma*sigma, -1.5));
      real3 tau  = make_real3(0.5/(sigma*sigma));
      
      sys->log<System::DEBUG2>("[BDHI::FCM] Particles to grid");
      /*Spread force on particles to grid positions -> S·F*/
      //Launch a small block per particle
      {
	int3 support = 2*P+1;
	int threadsPerParticle = 64;
	int numberNeighbourCells = support.x*support.y*support.z;
	if(numberNeighbourCells < 64) threadsPerParticle = 32;
	
	FCM_ns::particles2GridD<<<numberParticles, threadsPerParticle, 0, st>>>
	  (pos.raw(), v, d_gridVels, numberParticles, P, grid, prefactorGaussian, tau);
      }

    }
    void FCM::convolveFourier(hipStream_t st){
      hipfftSetStream(cufft_plan_forward, st);
      hipfftSetStream(cufft_plan_inverse, st);

      auto d_gridVels = thrust::raw_pointer_cast(gridVelsFourier.data());
      auto d_gridVelsFourier = thrust::raw_pointer_cast(gridVelsFourier.data());
            
      sys->log<System::DEBUG2>("[BDHI::FCM] Taking grid to wave space");
      {
	/*Take the grid spreaded forces and apply take it to wave space -> FFTf·S·F*/
	auto cufftStatus =
	  hipfftExecR2C(cufft_plan_forward,
		       (hipfftReal*)d_gridVels,
		       (hipfftComplex*)d_gridVelsFourier);
	if(cufftStatus != HIPFFT_SUCCESS){
	  sys->log<System::CRITICAL>("[BDHI::FCM] Error in forward CUFFT");
	}
      }
      sys->log<System::DEBUG2>("[BDHI::FCM] Wave space convolution");
      {
	/*Scale the wave space grid forces, transforming in velocities -> B·FFT·S·F*/
	//Launch a 3D grid of threads, a thread per cell.
	//Only the second half of the cells in the innermost (x) coordinate need to be processed, the rest are redundant and not used by cufft.
      
	dim3 NthreadsCells = dim3(8,8,8);
	dim3 NblocksCells;
	{
	  int ncellsx = grid.cellDim.x/2+1;
	  NblocksCells.x= (ncellsx/NthreadsCells.x + ((ncellsx%NthreadsCells.x)?1:0));
	  NblocksCells.y= grid.cellDim.y/NthreadsCells.y + ((grid.cellDim.y%NthreadsCells.y)?1:0);
	  NblocksCells.z= grid.cellDim.z/NthreadsCells.z + ((grid.cellDim.z%NthreadsCells.z)?1:0);
	}

	real dV = grid.cellSize.x*grid.cellSize.y*grid.cellSize.z;
            
	FCM_ns::forceFourier2Vel<<<NblocksCells, NthreadsCells, 0, st>>>
	  ((cufftComplex3*) d_gridVelsFourier, //Input: FFT·S·F
	   (cufftComplex3*) d_gridVelsFourier, //Output: B·FFT·S·F
	   viscosity,
	   dV,
	   grid);
	//The sqrt(2*T/dt) factor needs to be here because far noise is summed to the M·F term.
	/*Add the stochastic noise to the fourier velocities if T>0 -> 1/√σ·√B·dWw */
	if(temperature > real(0.0)){
	  sys->log<System::DEBUG2>("[BDHI::FCM] Wave space brownian noise");
	  static ullint counter = 0; //Seed the rng differently each call
	  counter++;
	  sys->log<System::DEBUG2>("[BDHI::FCM] Wave space brownian noise");
	  real prefactor = sqrt(2*temperature/(dt*dV));
	  FCM_ns::fourierBrownianNoise<<<NblocksCells, NthreadsCells, 0, st>>>(
			//In: B·FFT·S·F -> Out: B·FFT·S·F + 1/√σ·√B·dWw 
			(cufftComplex3*)d_gridVelsFourier, 
			grid,
			prefactor, // 1/√σ· sqrt(2*T/dt),
			viscosity,
			seed, //Saru needs two seeds apart from thread id
			counter);
	}
      }
      sys->log<System::DEBUG2>("[BDHI::FCM] Going back to real space");
      {
	/*Take the fourier velocities back to real space ->  FFTi·(B·FFT·S·F + 1/√σ·√B·dWw )*/
	auto cufftStatus =
	  hipfftExecC2R(cufft_plan_inverse,
		       (hipfftComplex*)d_gridVelsFourier,
		       (hipfftReal*)d_gridVels);
	if(cufftStatus != HIPFFT_SUCCESS){
	  sys->log<System::CRITICAL>("[BDHI::FCM] Error in inverse CUFFT");
	}
      }


    }
    void FCM::interpolateParticles(real3 *Mv, hipStream_t st){
      sys->log<System::DEBUG2>("[BDHI::FCM] Grid to particles");	    
      int numberParticles = pg->getNumberParticles();
      auto pos = pd->getPos(access::location::gpu, access::mode::read);
      real3* d_gridVels = (real3*)thrust::raw_pointer_cast(gridVelsFourier.data());

      //Gaussian spreading/interpolation kernel parameters, s(r) = prefactor*exp(-tau*r2)           
      real3 prefactorGaussian = make_real3(pow(2*M_PI*sigma*sigma, -1.5));
      real3 tau  = make_real3(0.5/(sigma*sigma));

      
      int3 support = 2*P+1;
      int BLOCKSIZE = 128;
      int Nthreads = BLOCKSIZE<numberParticles?BLOCKSIZE:numberParticles;
      int Nblocks  =  numberParticles/Nthreads +  ((numberParticles%Nthreads!=0)?1:0); 

      /*Interpolate the real space velocities back to the particle positions ->
	Output: Mv = Mw·F + sqrt(2*T/dt)·√Mw·dWw = σ·St·FFTi·(B·FFT·S·F + 1/√σ·√B·dWw )*/
      FCM_ns::grid2ParticlesD<<<Nblocks, Nthreads, 0, st>>>
	(pos.raw(), Mv, d_gridVels,
	 numberParticles, P, grid, prefactorGaussian, tau);
    }

    
    template<typename vtype>
    void FCM::Mdot_far(real3 *Mv, vtype *v, hipStream_t st){
      sys->log<System::DEBUG1>("[BDHI::FCM] Computing MF wave space....");
      /*Clean gridVels*/
      {
	int ncells = grid.cellDim.x*grid.cellDim.y*grid.cellDim.z;
	int BLOCKSIZE = 128;
	int Nthreads = BLOCKSIZE<ncells?BLOCKSIZE:ncells;
	int Nblocks  =  ncells/Nthreads +  ((ncells%Nthreads!=0)?1:0); 

	sys->log<System::DEBUG2>("[BDHI::FCM] Setting vels to zero...");
	//Note that the same storage space is used for Fourier and real space
	//The real space is the only one that needs to be cleared.
	auto d_gridVels = (real3*)thrust::raw_pointer_cast(gridVelsFourier.data());
	fillWithGPU<<<Nblocks, Nthreads, 0, st>>>(d_gridVels,
						  make_real3(0), ncells);
      }

      spreadParticles(v, st);
      convolveFourier(st);
      interpolateParticles(Mv, st);      
    }

    void FCM::computeMF(real3* MF, hipStream_t st){
      sys->log<System::DEBUG1>("[BDHI::FCM] Computing MF....");
      auto force = pd->getForce(access::location::gpu, access::mode::read);
      Mdot<real4>(MF, force.raw(), st);
    }

    void FCM::computeBdW(real3* BdW, hipStream_t st){
      //This part is included in Fourier space when computing MF
    }

    void FCM::computeDivM(real3* divM, hipStream_t st){}


    void FCM::finish_step(hipStream_t st){
      sys->log<System::DEBUG2>("[BDHI::FCM] Finishing step");
 
    }
  }
}
