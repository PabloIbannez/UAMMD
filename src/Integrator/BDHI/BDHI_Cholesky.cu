#include "hip/hip_runtime.h"
/*Raul P. Pelaez. 2017. Cholesky BDHI submodule implementation

  Computes the hydrodynamic interactions between particles in the system by
  maintaining a 3Nx3N mobility matrix with the RPY tensor in memory
  and explicitly computing M·F as a matrix-vector product.

  Note that only the upper part of M is stored, as M is symmetric.

  The brownian noise is computed as BdW = chol(M)·dW with cuSOLVER and cuBLAS

References:
[1] https://github.com/RaulPPelaez/UAMMD/wiki/BDHI_Cholesky
[2] https://github.com/RaulPPelaez/UAMMD/wiki/NBody-Forces
*/
#include"BDHI_Cholesky.cuh"
#include"misc/Transform.cuh"
#include"Interactor/NBodyForces.cuh"
#include<fstream>
using namespace std;
using namespace BDHI;

namespace Cholesky_ns{
/*Fills the 3Nx3N Mobility matrix with Mij = RPY(|rij|)(I-r^r) in blocks of 3x3 matrices*/
  __global__ void fillMobilityRPYD(real * __restrict__ M,
				   const  real4* __restrict__ R,
				   uint N,
				   real M0, BDHI::RPYUtils utils){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i>=N) return;
    uint n = 3*N;

    /*Self Diffusion*/
    
    for(int k = 0; k < 3; k++)
      for(int l = 0; l < 3; l++){
	M[3*i + k + n*(3*i + l)] =  real(0.0);
      }
    M[3*i + 0 + n*(3*i + 0)] = M0;
    M[3*i + 1 + n*(3*i + 1)] = M0;
    M[3*i + 2 + n*(3*i + 2)] = M0;
  
    real3 rij;
    real* rijp = &(rij.x);    
    real c1, c2;
    for(int j=i+1; j<N; j++){	
      rij = make_real3(R[j]) - make_real3(R[i]);
#ifdef SINGLE_PRECISION
      const real r = sqrtf(dot(rij, rij));
#else
      const real r = sqrt(dot(rij, rij));
#endif
      const real invr2 = 1.0/(r*r);
      /*Rotne-Prager-Yamakawa tensor */
      const real2 c12 = utils.RPY(r);      
      c1 = M0*c12.x;
      c2 = M0*c12.y*invr2;
      /*3x3 Matrix for each particle pair*/
      for(int k = 0; k < 3; k++)
 	for(int l = 0; l < 3; l++)
	  M[3*i + k + n*(3*j + l)] = c2*rijp[k]*rijp[l];
      /*Diagonal*/
      for(int k = 0; k<3; k++)  M[3*i + k + n*(3*j + k)] += c1;
    }
  }
}

Cholesky::Cholesky(real M0, real rh, int N):
  BDHI_Method(M0, rh, N), utilsRPY(rh), force3(N), M(3*N*3*N){
  cerr<<"\tInitializing Cholesky subsystem...";  
 
  M.memset(0);

  BLOCKSIZE = 128;
  Nthreads = BLOCKSIZE<N?BLOCKSIZE:N;
  Nblocks  =  N/Nthreads +  ((N%Nthreads!=0)?1:0); 

  /*Init cuSolver for BdW*/
  hipsolverDnCreate(&solver_handle);
  h_work_size = 0;//work size of operation

  cusolverDnpotrf_bufferSize(solver_handle, 
			     HIPBLAS_FILL_MODE_UPPER, 3*N, M.d_m, 3*N, &h_work_size);
  gpuErrchk(hipMalloc(&d_work, h_work_size));
  gpuErrchk(hipMalloc(&d_info, sizeof(int)));
  /*Init cuBLAS for MF*/ 
  status = hipblasCreate(&handle);
  if(status){cerr<<"ERROR with CUBLAS!!\n"<<endl; exit(1);}

  isMup2date = false;
  cerr<<"DONE!!"<<endl;  
}

  
Cholesky::~Cholesky(){
    hipblasDestroy(handle);
    hipFree(d_work);
    hipFree(d_info);
}


void Cholesky::setup_step(hipStream_t st){
  /*Fill the upper part of symmetric mobility matrix*/
  Cholesky_ns::fillMobilityRPYD<<<Nblocks, Nthreads, 0 ,st>>>(M.d_m, pos.d_m, N,
							      M0, utilsRPY);
  /*M contains the mobility tensor in this step*/
  isMup2date = true;
}

/*This little functor is used with Transform::transform and changes a real4 to a real3*/
// See https://github.com/RaulPPelaez/UAMMD/wiki/Utils
namespace Cholesky_ns{
  struct changetor{
    inline __device__ real3 operator()(real4 t) const{ return make_real3(t);}
  };
}


void Cholesky::computeMF(real3* MF, hipStream_t st){
  /*computeMF should be called before computeBdW*/
  static bool warning_printed = false;
  if(!isMup2date){
    setup_step(st);
    if(!warning_printed){
      cerr<<"WARNING!!: You should call computeMF inmediatly after setup_step, otherwise M will be compute twice per step!"<<endl;
      warning_printed = true;
    }
  }

  /*Morphs a real4 vector into a real3 one, needed by cublas*/
  hipblasSetStream(handle, st);
  Transform::transform<<<Nblocks, Nthreads, 0, st>>>(force.d_m,
						     force3.d_m,
						     Cholesky_ns::changetor(), N);
  
  real alpha = 1.0;
  real beta = 0;
  /*Compute M·F*/
  cublassymv(handle, HIPBLAS_FILL_MODE_UPPER,
	     3*N, 
	     &alpha,
	     M.d_m, 3*N,
	     (real*) force3.d_m, 1,
	     &beta,
	     (real*)MF, 1); 
}


void Cholesky::computeBdW(real3 *BdW, hipStream_t st){
  if(!isMup2date) setup_step();
  /*computeBdw messes up M, fills it with B*/
  isMup2date = false;
  hipsolverSetStream(solver_handle, st);
  
  /*Perform cholesky factorization, store B on LOWER part of M matrix*/
  cusolverDnpotrf(solver_handle, HIPBLAS_FILL_MODE_UPPER,
		  3*N, M.d_m, 3*N, d_work, h_work_size, d_info);
  hiprandSetStream(curng, st);
  /*Gen new noise in BdW*/
  hiprandGenerateNormal(curng, (real*) BdW, 3*N + ((3*N)%2), real(0.0), real(1.0));

  hipblasSetStream(handle, st);
  /*Compute B·dW -> y = M·y*/
  cublastrmv(handle, //B is an upper triangular matrix (with non unit diagonal)
	     HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
	     3*N,
	     M.d_m, 3*N,
	     (real*)BdW, 1);

}


namespace Cholesky_ns{
  /*Exactly the same as Lanczos_ns::divMTranverser.
    It is placed here for convinience when performing tests that involve 
    changing the input parameters to the class*/
  /*This Nbody Transverser computes the analytic divergence of the RPY tensor*/
  // https://github.com/RaulPPelaez/UAMMD/wiki/Nbody-Forces
  // https://github.com/RaulPPelaez/UAMMD/wiki/Transverser
  struct divMTransverser{
    divMTransverser(real3* divM, real M0, real rh): divM(divM), M0(M0), rh(rh){
      this->invrh = 1.0/rh;
    }
    
    inline __device__ real3 zero(){ return make_real3(real(0.0));}
    inline __device__ real3 compute(const real4 &pi, const real4 &pj){
      /*Work in units of rh*/
      const real3 r12 = (make_real3(pi)-make_real3(pj))*invrh;
      const real r2 = dot(r12, r12);
      if(r2==real(0.0))
	return make_real3(real(0.0));
#ifdef SINGLE_PRECISION
      real invr = rsqrtf(r2);
#else
      real invr = rsqrt(r2);
#endif
      /*Just the divergence of the RPY tensor in 2D, taken from A. Donev's notes*/
      /*The 1/6pia is in M0, the factor kT is in the integrator, and the factor 1/a is in set*/
      if(r2>real(4.0)){
	real invr2 = invr*invr;
	return real(0.75)*(r2-real(2.0))*invr2*invr2*r12*invr;
      }
      else{
	return real(0.09375)*r12*invr;
      }
    }
    inline __device__ void accumulate(real3 &total, const real3 &cur){total += cur;}
    
    inline __device__ void set(int id, const real3 &total){
      divM[id] = M0*total*invrh;
    }
  private:
    real3* divM;
    real M0;
    real rh, invrh;
  };

}

void Cholesky::computeDivM(real3* divM, hipStream_t st){
  /*A simple NBody transverser, see https://github.com/RaulPPelaez/UAMMD/wiki/NBody-Forces */
   Cholesky_ns::divMTransverser divMtr(divM, M0, utilsRPY.rh);
  
   NBodyForces<Cholesky_ns::divMTransverser> nbody_divM(divMtr, st);
  
   nbody_divM.sumForce();
}
 