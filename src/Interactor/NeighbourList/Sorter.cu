#include "Sorter.cuh"
#include<hipcub/hipcub.hpp>

namespace Sorter{

  void sortByKey(uint *&index, uint *&index_alt, uint *&hash, uint *&hash_alt, int N, hipStream_t st){
    //This uses the CUB API to perform a radix sort
    //CUB orders by key an array pair and copies them onto another pair
    //This function switches the arrays with _alt each time its called
    static bool init = false;
    static void *d_temp_storage = NULL;
    static size_t temp_storage_bytes = 0; //Additional storage needed by cub

    /**Initialize CUB at first call**/
    if(!init){
      /*Allocate temporal value/key pair*/
      gpuErrchk(hipMalloc(&hash_alt,  N*sizeof(uint)));
      gpuErrchk(hipMalloc(&index_alt, N*sizeof(uint)));    
      /*On first call, this function only computes the size of the required temporal storage*/
      hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
				      hash, hash_alt,
				      index, index_alt,
				      N,
				      0, sizeof(uint)*8,st);
      /*Allocate temporary storage*/
      gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
      init = true;
    }

    /**Perform the Radix sort on the index/hash pair**/
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
				    hash, hash_alt,
				    index, index_alt,
				    N,
				    0, sizeof(uint)*8,st);

    /**Swap the references**/
    std::swap(hash, hash_alt);
    std::swap(index, index_alt);

    // thrust::stable_sort_by_key(device_ptr<uint>(hash),
    // 			device_ptr<uint>(hash+N),
    // 			device_ptr<uint>(index));
    //cudaCheckErrors("Sort hash");					  


  }


}