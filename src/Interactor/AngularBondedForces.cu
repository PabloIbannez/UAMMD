#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2017


 */

/********************************THREE BONDED FORCES**********************************/

#include"AngularBondedForces.cuh"
#include"global/defines.h"
#include<iostream>
#include<fstream>
#include<vector>

namespace uammd{


  AngularBondedForces::~AngularBondedForces(){}


  AngularBondedForces::AngularBondedForces(shared_ptr<ParticleData> pd,
					   shared_ptr<System> sys,
					   Parameters par):
    Interactor(pd,
	       sys,
	       "AngularBondedForces"),
    TPP(64),
    box(par.box){
    
    int numberParticles = pg->getNumberParticles();
    sys->log<System::MESSAGE>("[AngularBondedForces] Initialized");

    nbonds = 0;
    /*Read the bond list from the file*/
    std::ifstream in(par.readFile);
    std::vector<std::vector<int>> isInBonds(numberParticles);
    in>>nbonds;
    std::vector<ThreeBond> blst(nbonds); //Temporal storage for the bonds in the file

  
    sys->log<System::MESSAGE>("[AngularBondedForces] Detected: %d particle-particle-particle bonds", nbonds);
    
    if(nbonds>0){
    
      for(int b=0; b<nbonds; b++){
	int i, j, k;
	in>>i>>j>>k;
	
	isInBonds[i].push_back(b);
	isInBonds[j].push_back(b);
	isInBonds[k].push_back(b);      
      
	blst[b].i = i;
	blst[b].j = j;
	blst[b].k = k;
      
	in>>blst[b].kspring>>blst[b].ang;
	
      }
    }
  
  
    bondList.resize(nbonds*3);//Allocate 3*nbonds
    bondStart.resize(numberParticles, 0xffFFffFF);
    bondEnd.resize(numberParticles, 0);

  
    fori(0, numberParticles){
      int nbondsi;
      nbondsi = isInBonds[i].size();
      if(nbondsi==0) continue;
    
      int offset;
      if(i>0)
	offset = bondEnd[i-1];
      else
	offset = 0;
    
      forj(0,nbondsi){
	bondList[offset+j] = blst[isInBonds[i][j]];
      }
      bondEnd[i] = offset+nbondsi;
      bondStart[i] = offset;
    }

    std::vector<int> pwb; //Particles with bonds
    fori(0, numberParticles){
      if(bondStart[i]!=0xffFFffFF){
	pwb.push_back(i);
      }
    }

    bondParticleIndex.assign(pwb.begin(), pwb.end());
  
    nbonds *= 3; //We store all the bonds in which every particle is involved, per particle.
   
    sys->log<System::MESSAGE>("[AngularBondedForces] %d particles are involved in at least one bond.",pwb.size());

  }




  namespace Bonded_ns{
    //Custom kernel to compute and sum the force in a three particle angle spring
    /*
      Computes the potential: V(theta) = 0.5 K(theta-theta_0)^2
      F(\vec{ri}) = d(V(theta))/d(cos(theta))·d(cos(theta))/d(\vec{ri})
    */
    __global__ void computeThreeBondedForce(real4* __restrict__ force,
					    const real4* __restrict__ pos,
					    const int* __restrict__ bondStart,
					    const int* __restrict__ bondEnd,
					    const int* __restrict__ bondedParticleIndex,
					    const AngularBondedForces::ThreeBond* __restrict__ bondList,
					    const int * __restrict__ id2index,
					    Box box){
      extern __shared__ real3 forceTotal[];
      /*A block per particle, as in computeBondedForcesD*/
      int p = id2index[bondedParticleIndex[blockIdx.x]];

      real3 posp = make_real3(pos[p]);
  
      int first = bondStart[p];
      int last = bondEnd[p];
   
      real3 f = make_real3(real(0.0));

      int i,j,k;             //The bond indices
      real3 posi,posj, posk; //The bond particles
      real kspring, ang0; //The bond info

      /*         i -------- j -------- k*/
      /*             rij->      <-rkj  */
    
      real3 rij, rkj; //rij = ri - rj
  
      real invsqrij, invsqrkj; //1/|rij|
      real rij2, rkj2;  //|rij|^2

    
      real a2; 
      real cijk, sijk;
      real a, a11, a12, a22;
      real ampli;

      /*Go through my bonds*/
      for(int b = first+threadIdx.x; b<last; b+=blockDim.x){
	/*Recover bond info*/
	auto bond = bondList[b];
	i = id2index[bond.i];
	j = id2index[bond.j];
	k = id2index[bond.k];

	kspring = bond.kspring;	
	ang0 = bond.ang;

      
	//TODO Texture memory target
	/*Store the positions of the three particles*/
	/*We already got one of them, p*/
	/*Differentiate between the three particles in the bond*/
	if(p==i){
	  posi = posp;
	  posj = make_real3(pos[j]);
	  posk = make_real3(pos[k]);
	}
	else if(p==j){
	  posi = make_real3(pos[i]);
	  posj = posp;
	  posk = make_real3(pos[k]);
	}
	else{
	  posi = make_real3(pos[i]);
	  posj = make_real3(pos[j]);
	  posk = posp;
	}

	/*Compute distances and vectors*/
	/***rij***/
	rij =  box.apply_pbc(posi-posj);
	rij2 = dot(rij, rij);
	invsqrij = rsqrt(rij2);
	/***rkj***/
	rkj =  box.apply_pbc(posk-posj);
	rkj2 = dot(rkj, rkj);
	invsqrkj = rsqrtf(rkj2);
	/********/
      
	a2 = invsqrij * invsqrkj;
	cijk = dot(rij, rkj)*a2; //cijk = cos (theta) = rij*rkj / mod(rij)*mod(rkj)

	/*Cos must stay in range*/
	if(cijk>real(1.0)) cijk = real(1.0);
	else if (cijk<real(-1.0)) cijk = -real(1.0);


	//Approximation for small angle displacements	
	//sijk = sqrt(real(1.0)-cijk*cijk); //sijk = sin(theta) = sqrt(1-cos(theta)^2)
	//sijk cant be zero to avoid division by zero
	//if(sijk<real(0.000001)) sijk = real(0.000001);
	//ampli = -kspring * (acosf(cijk) - ang0)/sijk; //The force amplitude -k·(theta-theta_0)

	
	if(ang0 == real(0.0) && false){
	  //TODO replace rij for rji so ang0=0 means straight and this can apply
	  //When ang0=pi means stragiht it is difficult to check if ang0 is pi
	  ampli = -kspring;
	}
	else{
	  const real theta = acosf(cijk);
	  const real sinthetao2 = sinf(real(0.5)*theta);
	  ampli = -kspring*(sintheto2- sinf(ang0*real(0.5)))/sinthetao2;
	}
	
	//ampli = -kang*(-sijk*cos(ang0)+cijk*sin(ang0))+ang0; //k(1-cos(ang-ang0))
	
	

	//Magical trigonometric relations to infere the direction of the force

	a11 = ampli*cijk/rij2;
	a12 = -ampli*a2;
	a22 = ampli*cijk/rkj2;
      
	/*Sum according to my position in the bond*/
	// i ----- j ------ k
	if(p==i){
	  f += make_real3(a11*rij + a12*rkj); //Angular spring	
	}
	else if(p==j){
	  //Angular spring
	  f -= make_real3(a11*rij + a12*rkj + a22*rkj + a12*rij);	
	}
	else if(p==k){
	  //Angular spring
	  f += make_real3(a22*rkj + a12*rij);
	}
      }

      //The fisrt thread sums all the contributions
      forceTotal[threadIdx.x] = f;
      __syncthreads();
      //TODO Implement a warp reduction
      if(threadIdx.x==0){
	real3 ft = make_real3(real(0.0));
	for(int i=0; i<blockDim.x; i++){
	  ft += forceTotal[i];
	}
	force[p] += make_real4(ft);
      }

    }



  }
  void AngularBondedForces::sumForce(hipStream_t st){
    if(nbonds>0){
      int Nparticles_with_bonds = bondParticleIndex.size();
      
      auto force = pd->getForce(access::location::gpu, access::mode::readwrite);
      auto pos = pd->getPos(access::location::gpu, access::mode::read);
      auto d_bondStart = thrust::raw_pointer_cast(bondStart.data());
      auto d_bondEnd = thrust::raw_pointer_cast(bondEnd.data());
      auto d_bondParticleIndex = thrust::raw_pointer_cast(bondParticleIndex.data());
      auto d_bondList = thrust::raw_pointer_cast(bondList.data());

      auto id2index = pd->getIdOrderedIndices(access::location::gpu);
      
      Bonded_ns::computeThreeBondedForce<<<Nparticles_with_bonds,
	TPP,
	TPP*sizeof(real3)>>>(force.raw(), pos.raw(),
			     d_bondStart,
			     d_bondEnd,
			     d_bondParticleIndex,
			     d_bondList,
			     id2index,
			     box);

    }

  }


  real AngularBondedForces::sumEnergy(){
    return 0;
  }

  // real AngularBondedForces::sumVirial(){
  //   return 0;
  // }

}
