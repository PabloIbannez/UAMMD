#include "hip/hip_runtime.h"

#include"BondedForcesGPU.cuh"
#include"utils/hip/hip_vector_types.h"
#include"utils/helper_gpu.cuh"
#include<thrust/device_ptr.h>
#include<thrust/reduce.h>
#include<thrust/for_each.h>
#include<thrust/iterator/zip_iterator.h>

using namespace thrust;

//Parameters in constant memory, super fast access
__constant__ BondedForcesParams bondedForcesParamsGPU; 



void initBondedForcesGPU(BondedForcesParams m_params){
  m_params.invL = 1.0f/m_params.L;
  /*Upload parameters to constant memory*/
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(bondedForcesParamsGPU), &m_params, sizeof(BondedForcesParams)));

}



//MIC algorithm
inline __device__ void apply_pbc(float3 &r){
  r -= floorf(r*bondedForcesParamsGPU.invL+0.5f)*bondedForcesParamsGPU.L; 
}
inline __device__ void apply_pbc(float4 &r){
  r -= floorf(r*bondedForcesParamsGPU.invL+0.5f)*bondedForcesParamsGPU.L; //MIC algorithm
}


//Thrust trick to apply a transformation to each element of an array, in parallel
struct bondedForces_functor{
  Bond   *bondList;
  float4 *d_pos;
  __host__ __device__ bondedForces_functor(Bond *bondList, float4* pos):
    bondList(bondList), d_pos(pos){}
  //The operation is performed on creation
  template <typename Tuple>
  __device__  void operator()(Tuple t){
    /*Retrive the data*/
    float4 pos = get<0>(t);
    float4 force = get<1>(t);
    uint first = get<2>(t); //bondStart
    uint last = get<3>(t);  //bondEnd

    /*If I am connected to some particle*/
    if(first!=0xffffffff){

      Bond bond;
      int j; float4 posj; //The other particle
      float r0, k; //The bond info
      
      float3 r12;
      /*For all particles connected to me*/
      for(int b = first; b<last; b++){
	/*Retrieve bond*/
	bond = bondList[b];
	j = bond.j;
	r0 = bond.r0;
	k = bond.k;
	posj = d_pos[j];

	/*Compute force*/
	r12 =  make_float3(pos-posj);
	apply_pbc(r12);

	float invr = rsqrt(dot(r12, r12));
	
	float fmod = -k*(1.0f-r0*invr); //F = -k·(r-r0)·rvec/r
	force += make_float4(fmod*r12);
      }
    }
    get<1>(t) = force;
  }
};


void computeBondedForce(float4 *force, float4 *pos,
			uint *bondStart, uint *bondEnd, Bond* bondList, uint N, uint nbonds){

  device_ptr<float4> d_pos4(pos);
  device_ptr<float4> d_force4(force);
  device_ptr<uint> d_bondStart(bondStart);
  device_ptr<uint> d_bondEnd(bondEnd);

  /**Thrust black magic to perform a multiple transformation, see the functor description**/
  for_each(
	   make_zip_iterator( make_tuple( d_pos4, d_force4, d_bondStart, d_bondEnd)),
	   make_zip_iterator( make_tuple( d_pos4 + N, d_force4 +N, d_bondStart+N, d_bondEnd+N)),
	   bondedForces_functor(bondList, pos)); 

}








struct bondedForcesFP_functor{
  BondFP   *bondListFP;
  __host__ __device__ bondedForcesFP_functor(BondFP *bondList):
    bondListFP(bondList){}
  //The operation is performed on creation
  template <typename Tuple>
  __device__  void operator()(Tuple t){
    /*Retrive the data*/
    float4 pos = get<0>(t);
    float4 force = get<1>(t);
    uint first = get<2>(t); //bondStart
    uint last = get<3>(t);  //bondEnd

    /*If I am connected to some particle*/
    if(first!=0xffffffff){

      BondFP bond;
      float4 posFP; //The other particle
      float r0, k; //The bond info
      
      float3 r12;
      /*For all particles connected to me*/
      for(int b = first; b<last; b++){
	/*Retrieve bond*/
	bond = bondListFP[b];
	r0 = bond.r0;
	k = bond.k;
	posFP = make_float4(bond.pos);

	/*Compute force*/
	r12 =  make_float3(pos-posFP);
	apply_pbc(r12);

	float invr = 0.0f;
	if(r0!=0.0f) invr = rsqrt(dot(r12, r12));
	
	float fmod = -k*(1.0f-r0*invr); //F = -k·(r-r0)·rvec/r
	force += make_float4(fmod*r12);
      }
    }
    get<1>(t) = force;
  }
};



void computeBondedForceFixedPoint(float4 *force, float4 *pos,
				  uint *bondStartFP, uint *bondEndFP, BondFP* bondListFP,
				  uint N, uint nbonds){

  device_ptr<float4> d_pos4(pos);
  device_ptr<float4> d_force4(force);
  device_ptr<uint> d_bondStart(bondStartFP);
  device_ptr<uint> d_bondEnd(bondEndFP);

  /**Thrust black magic to perform a multiple transformation, see the functor description**/
  for_each(
	   make_zip_iterator( make_tuple( d_pos4, d_force4, d_bondStart, d_bondEnd)),
	   make_zip_iterator( make_tuple( d_pos4 + N, d_force4 +N, d_bondStart+N, d_bondEnd+N)),
	   bondedForcesFP_functor(bondListFP)); 

}
