#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2016. Short range pair forces Interactor GPU callers and kernels.

  The Neighbour list is constructed in the GPU as follows:
  
  1-Compute a hash for each particle based on its cell. Store in particleHash, also fill particleIndex with the index of each particle (particleIndex[i] = i)
  2-Sort particleIndex based on particleHash (sort by key). This way the particles in a same cell are one after the other in particleIndex. The Morton hash also improves the memory acces patter in the GPU.
  3-Fill cellStart and cellEnd with the indices of particleIndex in which a cell starts and ends. This allows to identify where all the [indices of] particles in a cell are in particleIndex, again, one after the other.
  
  The transversal of this cell list is done by transversing, for each particle, the 27 neighbour cells of that particle's cell.
  

  Force is evaluated using table lookups (with texture memory)


  References:
  http://docs.nvidia.com/cuda/samples/5_Simulations/particles/doc/particles.pdf

  TODO:
  100- Implement many threads per particle in force compute
  100- Make number of blocks and threads to autotune
  100- Improve the transversing of the 27 neighbour cells
  90- Make energy measure custom for each potential, currently only LJ, hardcoded.
  90- Implement energy and virial compute in PairForcesDPD, maybe take it to another file
  80- General functions like apply_pbc should be made global to ease development.
  10- Find a way to properly handle the alternate arrays in sortCellIntex
*/

#include<hipcub/hipcub.hpp>
#include<hiprand/hiprand_kernel.h>
#include"PairForcesGPU.cuh"
#include"utils/hip/hip_vector_types.h"
#include"utils/helper_gpu.cuh"

#include<thrust/device_ptr.h>
#include<thrust/for_each.h>
#include<thrust/iterator/zip_iterator.h>
#include<thrust/sort.h>
#include<iostream>


typedef unsigned long long int ullint;

#define BLOCKSIZE 128

using namespace thrust;
using std::cerr;
using std::endl;

namespace pair_forces_ns{
  __constant__ Params params; //Simulation parameters in constant memory, super fast access
  __constant__ ParamsDPD paramsDPD; //Simulation parameters in constant memory, super fast access
  
  //  texture<float, 1, hipReadModeElementType> texForce; hipArray *dF;

  hipTextureObject_t h_texPos=0, h_texSortPos=0;
  hipTextureObject_t h_texCellStart=0, h_texCellEnd=0;
  hipTextureObject_t h_texVel=0, h_texSortVel=0;
  
  uint GPU_Nblocks;
  uint GPU_Nthreads;
  
  //Initialize gpu variables 
  void initPairForcesGPU(Params &m_params,
			 hipTextureObject_t texForce, hipTextureObject_t texEnergy,
			 uint *cellStart, uint *cellEnd, uint* particleIndex, uint ncells,
			 float4 *sortPos, float4 *pos, uint N){
    
    /*Precompute some inverses to save time later*/
    m_params.invrc2 = 1.0f/(m_params.rcut*m_params.rcut);
    m_params.invrc = 1.0f/(m_params.rcut);
    m_params.invL = 1.0f/m_params.L;
    m_params.invCellSize = 1.0f/m_params.cellSize;
    m_params.getCellFactor = make_float3(0.5f*m_params.L*m_params.invCellSize);
    m_params.gridPos2CellIndex = make_int3( 1,
					    m_params.cellDim.x,
					    m_params.cellDim.x*m_params.cellDim.y);
    
    /*Create texture objects*/
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = pos;
    resDesc.res.linear.desc = hipCreateChannelDesc<float4>();
    resDesc.res.linear.sizeInBytes = N*sizeof(float4);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(&h_texPos, &resDesc, &texDesc, NULL);


    resDesc.res.linear.devPtr = sortPos;
    hipCreateTextureObject(&h_texSortPos, &resDesc, &texDesc, NULL);

    resDesc.res.linear.devPtr = cellStart;
    resDesc.res.linear.desc = hipCreateChannelDesc<uint>();
    resDesc.res.linear.sizeInBytes = ncells*sizeof(uint);
    
    hipCreateTextureObject(&h_texCellStart, &resDesc, &texDesc, NULL);

    resDesc.res.linear.devPtr = cellEnd;
    hipCreateTextureObject(&h_texCellEnd, &resDesc, &texDesc, NULL);
        
    m_params.texForce =  texForce;
    m_params.texEnergy =  texEnergy;
    
    /*Upload parameters to constant memory*/
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(params), &m_params, sizeof(Params)));
    
    /*Each particle is asigned a thread*/
    GPU_Nthreads = BLOCKSIZE<N?BLOCKSIZE:N;
    GPU_Nblocks  =  N/GPU_Nthreads +  ((N%GPU_Nthreads!=0)?1:0); 
  }


  
  void initPairForcesDPDGPU(ParamsDPD &m_params, float4* sortVel, uint N){

    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(paramsDPD), &m_params, sizeof(ParamsDPD)));


    /*Create texture obsjects*/
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = sortVel;
    resDesc.res.linear.desc = hipCreateChannelDesc<float4>();
    resDesc.res.linear.sizeInBytes = N*sizeof(float4);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(&h_texSortVel, &resDesc, &texDesc, NULL);


    //gpuErrchk(hipBindTexture(NULL, texSortVel, sortVel, N*sizeof(float4)));
  }

  /****************************HELPER FUNCTIONS*****************************************/
  //MIC algorithm
  template<typename vecType>
  inline __device__ void apply_pbc(vecType &r){
    r -= floorf(r*params.invL+0.5f)*params.L; //MIC Algorithm
  }

  //Get the 3D cell p is in, just pos in [0,L] divided by ncells(vector) .INT DANGER.
  template<typename vecType>
  inline __device__ int3 getCell(vecType p){
    apply_pbc(p); //Reduce to MIC
    // return  int( (p+0.5L)/cellSize )
    int3 cell = make_int3((p+0.5f*params.L)*params.invCellSize);
    //Anti-Traquinazo guard, you need to explicitly handle the case where a particle
    // is exactly at the box limit, AKA -L/2. This is due to the precision loss when
    // casting int from floats, which gives non-correct results very near the cell borders.
    // This is completly neglegible in all cases, except with the cell 0, that goes to the cell
    // cellDim, wich is catastrophic.
    //Doing the previous operation in double precision (by changing 0.5f to 0.5) also works, but it is a bit of a hack and the performance appears to be the same.
    if(cell.x==params.cellDim.x) cell.x = 0;
    if(cell.y==params.cellDim.y) cell.y = 0;
    if(cell.z==params.cellDim.z) cell.z = 0;
    return cell;
  }

  //Apply pbc to a cell coordinates
  inline __device__ void pbc_cell(int3 &cell){
    if(cell.x==-1) cell.x = params.cellDim.x-1;
    else if(cell.x==params.cellDim.x) cell.x = 0;

    if(cell.y==-1) cell.y = params.cellDim.y-1;
    else if(cell.y==params.cellDim.y) cell.y = 0;

    if(cell.z==-1) cell.z = params.cellDim.z-1;
    else if(cell.z==params.cellDim.z) cell.z = 0;
  }
  
  //Get linear index of a 3D cell, from 0 to ncells-1
  inline __device__ uint getCellIndex(int3 gridPos){
    return dot(gridPos, params.gridPos2CellIndex);
  }


  /*Interleave a 10 bit number in 32 bits, fill one bit and leave the other 2 as zeros.*/
  inline __device__ uint encodeMorton(uint i){
  
    uint x = i;
  
    x &= 0x3ff;
    x = (x | x << 16) & 0x30000ff;
    x = (x | x << 8) & 0x300f00f;
    x = (x | x << 4) & 0x30c30c3;
    x = (x | x << 2) & 0x9249249;
    return x;
  }
  /*Fuse three 10 bit numbers in 32 bits, producing a Z order Morton hash*/
  inline __device__ uint mortonHash(const int3 &cell){

    return encodeMorton(cell.x) | (encodeMorton(cell.y) << 1) | (encodeMorton(cell.z) << 2);
  }
  
  /****************************************************************************************/


  /*Assign a hash to each particle from its cell index*/
  __global__ void calcHashD(uint __restrict__ *particleHash, uint __restrict__ *particleIndex, 
			    const float4 __restrict__ *pos, uint N){
    uint index = blockIdx.x*blockDim.x + threadIdx.x;  
    if(index>=N) return;
    float4 p = pos[index];

    int3 cell = getCell(p);
    /*The particleIndex array will be sorted by the hashes, any order will work*/
    uint hash = mortonHash(cell);//getCellIndex(cell);
    /*Before ordering by hash the index in the array is the index itself*/
    particleIndex[index] = index;
    particleHash[index]  = hash;
  }  
  //CPU kernel caller
  void calcHash(float4 *pos, uint *particleHash, uint *particleIndex, uint N){
    calcHashD<<<GPU_Nblocks, GPU_Nthreads>>>(particleHash, particleIndex, pos, N);
    //cudaCheckErrors("Calc hash");					   
  }



  /*Sort the particleIndex list by hash*/
  // this allows to access the neighbour list of each particle in a more coalesced manner
  //Each time this is called, the pointers particleHash and particleIndex are swapped
  void sortCellHash(uint *&particleHash, uint *&particleIndex, uint N){
    //This uses the CUB API to perform a radix sort
    //CUB orders by key an array pair and copies them onto another pair
    //This function stores an internal key/value pair and switches the arrays each time
    static bool init = false;
    static void *d_temp_storage = NULL;
    static size_t temp_storage_bytes = 0; //Additional storage needed by cub
    static uint *particleHash_alt = NULL, *particleIndex_alt = NULL; //Additional key/value pair

    static hipcub::DoubleBuffer<uint> d_keys;
    static hipcub::DoubleBuffer<uint> d_values;
    /**Initialize CUB at first call**/
    if(!init){
      /*Allocate temporal value/key pair*/
      gpuErrchk(hipMalloc(&particleHash_alt, N*sizeof(uint)));
      gpuErrchk(hipMalloc(&particleIndex_alt, N*sizeof(uint)));
    
      /*Create this CUB like data structure*/
      d_keys = hipcub::DoubleBuffer<uint>(particleHash, particleHash_alt);    
      d_values = hipcub::DoubleBuffer<uint>(particleIndex, particleIndex_alt);
      /*On first call, this function only computes the size of the required temporal storage*/
      hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
     				      d_keys, 
     				      d_values, N);
      /*Allocate temporary storage*/
      gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
      init = true;
    }

    /**Perform the Radix sort on the index/cell pair**/
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
     				    d_keys, 
     				    d_values, N); 
    /**Switch the references**/
    particleHash     = d_keys.Current();
    particleIndex = d_values.Current();

    particleHash_alt     = d_keys.Alternate();
    particleIndex_alt = d_values.Alternate();

    // thrust::stable_sort_by_key(device_ptr<uint>(particleHash),
    // 			device_ptr<uint>(particleHash+N),
    // 			device_ptr<uint>(particleIndex));
    //cudaCheckErrors("Sort hash");					   
  }

  /*This kernel fills sortPos with the positions in pos, acording to the indices in particleIndex*/
  __global__ void reorderPosD(float4 *sortPos,
			      hipTextureObject_t texPos,
			      const uint* __restrict__ particleIndex, uint N){
    uint i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i>=N) return;

    uint sort_index = particleIndex[i]; //Coalesced

    sortPos[i] = tex1Dfetch<float4>(texPos, sort_index);
  }
  /*Same as above, but reordering vel aswell*/
  __global__ void reorderPosVelD(float4 *sortPos,
				 hipTextureObject_t texPos,
				 float4* sortVel,
				 float3 * vel,
				 uint* particleIndex, uint N){
    uint i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i>=N) return;

    uint sort_index = particleIndex[i]; //Coalesced

    sortPos[i] = tex1Dfetch<float4>(texPos, sort_index);
    //    sortVel[i] = tex1Dfetch<float4>(texVel, sort_index);
    sortVel[i] = make_float4(vel[sort_index]);
  }
  
  /*Fill CellStart and CellEnd*/
  __global__ void fillCellListD(const float4 __restrict__ *sortPos,
				uint *cellStart, uint *cellEnd,
				uint N){
    /*A thread per particle*/
    uint i = blockIdx.x*blockDim.x + threadIdx.x;

    if(i<N){//If my particle is in range
      uint icell, icell2;
      /*Get my icell*/
      icell = getCellIndex(getCell(sortPos[i]));
      /*Get the previous part.'s icell*/
      if(i>0){ /*Shared memory target VVV*/
	icell2 = getCellIndex(getCell(sortPos[i-1]));
      }
      else
	icell2 = 0;
      //If my particle is the first or is in a different cell than the previous
      //my i is the start of a cell
      if(i ==0 || icell != icell2){
	//Then my particle is the first of my cell
	cellStart[icell] = i;
	//If my i is the start of a cell, it is also the end of the previous
	//Except if I am the first one
	if(i>0)
	  cellEnd[icell2] = i;
      }
      //If I am the last particle my cell ends 
      if(i == N-1) cellEnd[icell] = N;

      
    }

  }

  /*Reupload the parameters to constant memory*/
  void updateParams(Params m_params){
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(params), &m_params, sizeof(Params)));
  }

  /*Create the Cell List from scratch in the GPU*/
  void makeCellList(float4 *pos, float4 *sortPos,
		    uint *&particleIndex, uint *&particleHash,
		    uint *cellStart, uint *cellEnd,
		    uint N, uint ncells){
    
    hipMemset(cellStart, 0xffffffff, ncells*sizeof(uint));

    calcHashD<<<GPU_Nblocks, GPU_Nthreads>>>(particleHash, particleIndex, pos, N);
    
    sortCellHash(particleHash, particleIndex, N);

    reorderPosD<<<GPU_Nblocks, GPU_Nthreads>>>(sortPos, h_texPos, particleIndex, N);

    /*This fills cellStart and cellEnd*/
    fillCellListD<<<GPU_Nblocks, GPU_Nthreads>>>(sortPos, cellStart, cellEnd, N);
    
  }
  void makeCellListDPD(float4 *pos, float3* vel,  float4 *sortPos, float4 *sortVel,
		       uint *&particleIndex, uint *&particleHash,
		       uint *cellStart, uint *cellEnd,
		       uint N, uint ncells){
    
    hipMemset(cellStart, 0xffffffff, ncells*sizeof(uint));

    calcHashD<<<GPU_Nblocks, GPU_Nthreads>>>(particleHash, particleIndex, pos, N);
    
    sortCellHash(particleHash, particleIndex, N);

    reorderPosVelD<<<GPU_Nblocks, GPU_Nthreads>>>(sortPos, h_texPos, sortVel, vel, particleIndex, N);

    fillCellListD<<<GPU_Nblocks, GPU_Nthreads>>>(sortPos, cellStart, cellEnd, N);
    
  }
		    
		    



  //TODO The naming and explanation of this function
  /*Transverses all the neighbour particles of each particle using the cell list and computes a quantity as implemented by Transversable. Each thread goes through all the neighbours of a certain particle(s)(index), transversing its 27 neighbour cells*/
  /*Computes a quantity determined by Transversable, which is a class that must implement the following methods:
    zero() -> returns the initial value of the quantity, in whatever type
    compute(float4 r1, float4 r2) -> compute the quantity depending of the pair positions/types
    set(uint index, TYPE quantity) -> sum the total quantity on particle index to global memory

    This quantity can be i.e a float4 and compute the force
                         or a float and compute the energy...
    */
  template<class Transversable>
  __global__ void transverseListD(Transversable T, 
				  hipTextureObject_t texSortPos,
				  const uint* __restrict__ particleIndex,
				  hipTextureObject_t texCellStart, hipTextureObject_t texCellEnd,
				  uint N){
    uint ii =  blockIdx.x*blockDim.x + threadIdx.x;

    //Grid-stride loop
    for(int index = ii; index<N; index += blockDim.x * gridDim.x){
      /*Compute force acting on particle particleIndex[index], index in the new order*/
      float4 pos = tex1Dfetch<float4>(texSortPos, index);

      /*Initial value of the quantity*/
      auto quantity = T.zero();
      
      int3 celli = getCell(pos);

      int x,y,z;
      int3 cellj;
      float4 posj;
      /**Go through all neighbour cells**/
      //For some reason unroll doesnt help here
      for(z=-1; z<=1; z++)
	for(y=-1; y<=1; y++)
	  for(x=-1; x<=1; x++){
	    cellj = celli + make_int3(x,y,z);
	    pbc_cell(cellj);

	    uint icell  = getCellIndex(cellj);
	    /*Index of the first particle in the cell's list*/ 
	    uint firstParticle = tex1Dfetch<uint>(texCellStart, icell);
	    /*Index of the last particle in the cell's list*/
	    uint lastParticle = lastParticle=tex1Dfetch<uint>(texCellEnd, icell);
	    // if(firstParticle!=0xffFFffFF)  
	    // else continue;
	    /*Because the list is ordered, all the particle indices in the cell are coalescent!*/
	    /*If there are no particles in the cell, firstParticle=0xffffffff, the loop is not computed*/
	    /*The fetch of lastParticle eitherway reduces branch divergency and is actually faster than checking
	      firstParticle before fetching*/
	    
	    for(uint j=firstParticle; j<lastParticle; j++){
	      /*Retrieve j pos*/
	      posj = tex1Dfetch<float4>(texSortPos, j);
	      /*Add force, i==j is handled in forceij */
	      quantity += T.compute(pos, posj);      
	    }
	    
	  }
      /*Write quantity with the original order*/
      uint pi = particleIndex[index]; //Coalesced
      T.set(pi, quantity);
    }
    
  }
    


  /***************************************FORCE*****************************/
  
  //tags: force compute force function forceij
  /*A helper class that holds the address of the force array (in device) and
    computes the force between two particles*/
  /*It also updates the global array newForce with the total force acting on particle pi*/
  /*This helper class can be passed as an argument to transverseListD, which will apply the compute
    function to every neighbouring particle pair in the system*/
  /*In order to compute any other quantity create a class like this, implementing the same functions
    but with any desired type, instead of float4 as in this case*/
  class forceTransversable{
  public:
    /*I need the device pointer to force*/
    forceTransversable(float4 *newForce):newForce(newForce){
    };
    /*Compute the force between two positions*/
    inline __device__ float4 compute(const float4 &R1,const float4 &R2){
      
      float3 r12 = make_float3(R2-R1);
      apply_pbc(r12);

      /*Squared distance*/
      float r2 = dot(r12,r12);
      /*Squared distance between 0 and 1*/
      float r2c = r2*params.invrc2;
      /*Both cases handled in texForce*/
      /*Check if i==j. This way reduces warp divergence and its faster than checking i==j outside*/
      //if(r2c==0.0f) return make_float4(0.0f);  
      /*Beyond rcut..*/
      //else if(r2c>=1.0f) return make_float4(0.0f);
      /*Get the force from the texture*/
      //float fmod = tex1D(texForce, r2c);
      float fmod = tex1D<float>(params.texForce, r2c);
      // float invr2 = 1.0f/r2;
      // float invr6 = invr2*invr2*invr2;
      // float invr8 = invr6*invr2;
      // //float E =  2.0f*invr6*(invr6-1.0f);
      // float fmod = -48.0f*invr8*invr6+24.0f*invr8;
      return  make_float4(fmod*r12);
    }
    /*Update the force acting on particle pi, pi is in the normal order*/
    inline __device__ void set(uint pi, const float4 &totalForce){
      newForce[pi] += totalForce;
    }
    /*Initial value of the force, this is a trick to allow the template in transverseList
      to guess the type of my quantity, a float4 in this case. Just set it to the 0 value of 
    the type of your quantity (0.0f for a float i.e)*/
    inline __device__ float4 zero(){
      return make_float4(0.0f);
    }
  private:
    float4* newForce;
  };


  
  //CPU kernel caller
  void computePairForce(float4 *sortPos, float4 *force,
			uint *cellStart, uint *cellEnd,
			uint *particleIndex, 
			uint N){
    /*An instance of the class that holds the function that computes the force*/
    forceTransversable ft(force); //It needs the addres of the force in device memory
    /*Transverse the neighbour list for each particle, using ft to compute the force in each pair*/
    transverseListD<<<GPU_Nblocks, GPU_Nthreads>>>(ft, h_texSortPos,
						   particleIndex,
						   h_texCellStart, h_texCellEnd,
						   N);
    //cudaCheckErrors("computeForce");
  }

  /****************************ENERGY***************************************/

  /*This class is analogous to forceTransversable, see for reference*/
  //tags: energy compute energyij
  class energyTransversable{
  public:
    energyTransversable(float *Energy):Energy(Energy){ };
    /*Returns the energy between two positions*/
    inline __device__ float compute(const float4 &R1,const float4 &R2){
      float3 r12 = make_float3(R2-R1);

      apply_pbc(r12);

      float r2 = dot(r12,r12);
      /*Squared distance between 0 and 1*/
      float r2c = r2*params.invrc2;
      /*Check if i==j. This way reduces warp divergence and its faster than checking i==j outside*/
      //if(r2c==0.0f) return 0.0f;  //Both cases handled in texForce
      /*Beyond rcut..*/
      //else if(r2c>=1.0f) return 0.0f;
      /*Get the force from the texture*/
      //float fmod = tex1D(texForce, r2c);
      //float invr2 = 1.0f/r2;
      //float invr6 = invr2*invr2*invr2;
      //TODO take from a texture*/
      //float E =  2.0f*invr6*(invr6-1.0f);
      float E = tex1D<float>(params.texEnergy, r2c);
      
      return E;
    }
    inline __device__ void set(uint pi, float energy){
      Energy[pi] = energy;
    }
    inline __device__ float zero(){
      return 0.0f;
    }
  private:
    float *Energy;
  };

  
  float computePairEnergy(float4 *sortPos, float *energy,
			  uint *cellStart, uint *cellEnd,
			  uint *particleIndex, 
			  uint N){

    /*Analogous to computeForce, see for reference*/
    energyTransversable et(energy);
    transverseListD<<<GPU_Nblocks, GPU_Nthreads>>>(et, h_texSortPos,
						   particleIndex,
						   h_texCellStart, h_texCellEnd,
						   N);
    device_ptr<float> d_e(energy);
    float sum;
    sum = thrust::reduce(d_e, d_e+N, 0.0f);
    //Returns energy per particle*/
    return (sum/(float)N);
    //cudaCheckErrors("computeForce");
  }


  /****************************VIRIAL***************************************/
  /*Analogous to forceTransversable, see for reference*/
  //tags: virial compute virialij
  class virialTransversable{
  public:
    virialTransversable(float *virial):Virial(virial){ };
    inline __device__ float compute(const float4 &R1,const float4 &R2){
      float3 r12 = make_float3(R2-R1);
      apply_pbc(r12);

      float r2 = dot(r12,r12);
      /*Squared distance between 0 and 1*/
      float r2c = r2*params.invrc2;
      //if(r2c==0.0f) return 0.0f; //No need to check i==j, tex1D(texForce, 0.0) = 0.0
      /*Beyond rcut..*/
      //if(r2c>=1.0f) return 0.0f; //Also 0 in texForce
      /*Get the force from the texture*/
      float fmod = tex1D<float>(params.texForce, r2c);
      // P = rhoKT + (1/2dV)sum_ij( Fij·rij ) //Compute only the Fij·rij, the rest is done outside
      return dot(fmod*r12,r12);
    }
    inline __device__ void set(uint pi, float virial){
      Virial[pi] = virial;
    }
    inline __device__ float zero(){
      return 0.0f;
    }
  private:
    float *Virial;
  };




  //CPU kernel caller
  float computePairVirial(float4 *sortPos, float *virial,
			  uint *cellStart, uint *cellEnd,
			  uint *particleIndex, 
			  uint N){

    virialTransversable ft(virial);
    transverseListD<<<GPU_Nblocks, GPU_Nthreads>>>(ft, h_texSortPos,
						   particleIndex,
						   h_texCellStart, h_texCellEnd,
						   N);
    device_ptr<float> d_vir(virial);
    float sum;
    // P = rhoKT + (1/2dV)sum_ij( Fij·rij ) This function returns (1/2)sum_ij( Fij·rij )
    sum = thrust::reduce(d_vir, d_vir+N, 0.0f);
    return (sum/2.0f);
    //cudaCheckErrors("computeForce");
  }



  /*******************************************DPD********************************************/

  /**********************FORCE********************/


  //Random number, the seed is used to recover a certain number in the random stream
  inline __device__ float randGPU(const ullint &seed, hiprandState *rng){
    hiprand_init(seed, 0, 0, rng);
    return hiprand_normal(rng);
  }


  //Computes the force between to positions
  inline __device__ float4 forceijDPD(const float4 &R1,const float4 &R2,
				      const float4 &V1,const float4 &V2, const float &randij){
  
    float3 r12 = make_float3(R1-R2);
    float3 v12 = make_float3(V1-V2);
  
    apply_pbc(r12);

    float r2 = dot(r12,r12);
    /*Squared distance between 0 and 1*/
    float r2c = r2*params.invrc2;
  
    float fmod= 0.0f;
  
    float w = 0.0f; //The intensity of the DPD thermostat 
    float rinv = 0.0f;
    if(r2c<1.0f){
      if(r2c==0.0f) return make_float4(0.0f);
      //w = r-rc -> linear
      rinv = rsqrt(r2);
      w = rinv-params.invrc;
    }
    else return make_float4(0.0f);
    //fmod = paramsDPD.A*w; //Soft force
  
    fmod -= tex1D<float>(params.texForce, r2c); //Conservative force
    fmod -= paramsDPD.gamma*w*w*dot(r12,v12); //Damping
    fmod += paramsDPD.noiseAmp*randij*w; //Random force
    return make_float4(fmod*r12);
  }

  //Computes the force acting on particle index from particles in cell cell
  inline __device__ float4 forceCellDPD(const int3 &cell, const uint &index,
					const float4 &pos, hipTextureObject_t texSortPos,
					const float4 &veli, hipTextureObject_t texSortVel,
					uint N,
					hiprandState &rng, const ullint &seed,
					hipTextureObject_t texCellStart,hipTextureObject_t texCellEnd){
    uint icell  = getCellIndex(cell);
    /*Index of the first particle in the cell's list*/ 
    uint firstParticle = tex1Dfetch<uint>(texCellStart, icell);

    float4 force = make_float4(0.0f);
    float4 posj, velj;
  
    /*Index of the last particle in the cell's list*/
    uint lastParticle = tex1Dfetch<uint>(texCellEnd, icell);
    /*Because the list is ordered, all the particle indices in the cell are coalescent!*/
    /*If there are no particles in the cell, firstParticle=0xffffffff, the loop is not computed*/
    /*The fetch of lastParticle eitherway reduces branch divergency and is actually faster than checking
      firstParticle before fetching*/
    float randij;
    ullint i0, j0;
    for(uint j=firstParticle; j<lastParticle; j++){
      /*Retrieve j pos and vel*/
      posj = tex1Dfetch<float4>(texSortPos, j);
      velj = tex1Dfetch<float4>(texSortVel, j);
      /*Prepare the seed for the RNG, it must be the same seed
	for pair ij and ji!*/
      if(index<j){
	i0=index;
	j0=j;
      }
      else{
	i0=j;
	j0=index;
      }
      /*Get the random number*/
      randij = randGPU(i0+(ullint)N*j0 +seed, &rng);
      /*Sum the force*/
      force += forceijDPD(pos, posj, veli, velj, randij);
    }
  
    return force;
  }


  //Kernel to compute the force acting on all particles
  __global__ void computeForceDDPD(hipTextureObject_t texSortPos, hipTextureObject_t texSortVel,
				   const uint __restrict__ *particleIndex,
				   hipTextureObject_t texCellStart, hipTextureObject_t texCellEnd,
				   float4* __restrict__ newForce,
				   uint N, ullint seed){
    /*Travel the particles per sort order*/
    uint ii =  blockIdx.x*blockDim.x + threadIdx.x;
    hiprandState rng;
  
    //Grid-stride loop
    for(int index = ii; index<N; index += blockDim.x * gridDim.x){
      /*Compute force acting on particle particleIndex[index], index in the new order*/
      float4 pos = tex1Dfetch<float4>(texSortPos, index);
      float4 veli= tex1Dfetch<float4>(texSortVel, index);
      //float3 veli = vel[pi];
      float4 force = make_float4(0.0f);
      int3 celli = getCell(pos);

      int x,y,z;
      int3 cellj;
      /**Go through all neighbour cells**/
      //For some reason unroll doesnt help here
      for(z=-1; z<=1; z++)
	for(y=-1; y<=1; y++)
	  for(x=-1; x<=1; x++){
	    cellj = celli+make_int3(x,y,z);
	    pbc_cell(cellj);	
	    force += forceCellDPD(cellj, index, pos, texSortPos, veli, texSortVel, N, rng, seed, texCellStart, texCellEnd);
	  }
      /*Write force with the original order*/
      uint pi = particleIndex[index]; 
      newForce[pi] += force;
    }
  }

  //CPU kernel caller
  void computePairForceDPD(float4 *force,
			   uint *particleIndex,
			   uint N, ullint seed){
    computeForceDDPD<<<GPU_Nblocks, GPU_Nthreads>>>(h_texSortPos, h_texSortVel,
						    particleIndex,
						    h_texCellStart,h_texCellEnd,
      
						    force,
						    N, seed);

    //cudaCheckErrors("computeForce");
  }

}
  
