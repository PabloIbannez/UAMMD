#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2016. Short range pair forces Interactor GPU callers and kernels.

Functions to compute the force acting on each particle

Neighbour list GPU implementation using hash short with cell index as hash.

References:
http://docs.nvidia.com/cuda/samples/5_Simulations/particles/doc/particles.pdf

TODO:
100- Use Z-order curve as hash instead of cell index to improve memory coherence when traveling the neighbour cells
90- Add support for particle types, encode in pos.w
90- Make energy measure custom for each potential, currently only LJ, hardcoded.
50- Try bindless textures again.
40- pbc_cells could be done better, this could improve force compute
10- There is no need to reconstruct the neighbour list from scratch each step,
  although computing the force is 50 times as expensive as this right now.
10- Find a way to properly handle the alternate arrays in sortCellIntex
*/

#include<cub/hipcub/hipcub.hpp>
#include<hiprand/hiprand_kernel.h>
#include"PairForcesGPU.cuh"
#include"utils/hip/hip_vector_types.h"
#include"utils/helper_gpu.cuh"

#include<thrust/device_ptr.h>
#include<thrust/for_each.h>
#include<thrust/iterator/zip_iterator.h>
#include<thrust/sort.h>
#include<iostream>


typedef unsigned long long int ullint;

#define BLOCKSIZE 128

using namespace thrust;
using std::cerr;
using std::endl;

__constant__ PairForcesParams params; //Simulation parameters in constant memory, super fast access
__constant__ PairForcesParamsDPD paramsDPD; //Simulation parameters in constant memory, super fast access

//Texture references for scattered access
texture<uint> texCellStart, texCellEnd, texParticleIndex;
texture<float4> texSortPos;
texture<float4> texSortVel; //For DPD
texture<float, 1, hipReadModeElementType> texForce; hipArray *dF;
//texture<float,1 , hipReadModeElementType> texEnergy; hipArray *dE;


uint GPU_Nblocks;
uint GPU_Nthreads;


//Initialize gpu variables 
void initPairForcesGPU(PairForcesParams m_params,
		       float *potForceData, float *potEnergyData, size_t potSize,
		       uint *cellStart, uint *cellEnd, uint* particleIndex, uint ncells,
		       float4 *sortPos, uint N){

  /*Precompute some inverses to save time later*/
  m_params.invrc2 = 1.0f/(m_params.rcut*m_params.rcut);
  m_params.invrc = 1.0f/(m_params.rcut);
  m_params.invL = 1.0f/m_params.L;
  m_params.invCellSize = 1.0f/m_params.cellSize;
  m_params.getCellFactor = 0.5f*m_params.L*m_params.invCellSize;
  
  /*Texture bindings, these ones are accessed by element*/ 
  gpuErrchk(hipBindTexture(NULL, texCellStart, cellStart, ncells*sizeof(uint)));
  gpuErrchk(hipBindTexture(NULL, texCellEnd,   cellEnd,   ncells*sizeof(uint)));
  gpuErrchk(hipBindTexture(NULL, texParticleIndex,   particleIndex,   (N+1)*sizeof(uint)));
  gpuErrchk(hipBindTexture(NULL, texSortPos, sortPos, N*sizeof(float4)));

  /*Create and bind force texture, this needs interpolation*/
  hipChannelFormatDesc channelDesc;
  channelDesc = hipCreateChannelDesc(32, 0,0,0, hipChannelFormatKindFloat);

  gpuErrchk(hipMallocArray(&dF,
			    &channelDesc,
			    potSize/sizeof(float),1));

  gpuErrchk(hipMemcpyToArray(dF, 0,0, potForceData, potSize, hipMemcpyHostToDevice));

  texForce.normalized = true; //The values are fetched between 0 and 1
  texForce.addressMode[0] = hipAddressModeClamp; //0 outside [0,1]
  texForce.filterMode = hipFilterModeLinear; //Linear filtering

  /*Texture binding*/
  gpuErrchk(hipBindTextureToArray(texForce, dF, channelDesc));
  
  /**SAME WITH THE ENERGY**/
  // gpuErrchk(hipMallocArray(&dE,
  // 			    &channelDesc,
  // 			    potSize/sizeof(float),1));
  // gpuErrchk(hipMemcpyToArray(dE, 0,0, potEnergyData, potSize, hipMemcpyHostToDevice));
  // /*Texture binding*/
  // gpuErrchk(hipBindTextureToArray(texEnergy, dE, channelDesc));


  /*Upload parameters to constant memory*/
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(params), &m_params, sizeof(PairForcesParams)));



  /*Each particle is asigned a thread*/
  GPU_Nthreads = BLOCKSIZE<N?BLOCKSIZE:N;
  GPU_Nblocks  =  N/GPU_Nthreads +  ((N%GPU_Nthreads!=0)?1:0); 
}



void initPairForcesDPDGPU(PairForcesParamsDPD m_params, float4* sortVel, uint N){

  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(paramsDPD), &m_params, sizeof(PairForcesParamsDPD)));


  gpuErrchk(hipBindTexture(NULL, texSortVel, sortVel, N*sizeof(float4)));
}

/****************************HELPER FUNCTIONS*****************************************/
//MIC algorithm
inline __device__ void apply_pbc(float3 &r){
  r -= floorf(r*params.invL+0.5f)*params.L; 
}
inline __device__ void apply_pbc(float4 &r){
  r -= floorf(r*params.invL+0.5f)*params.L; //MIC algorithm
}

//Get the 3D cell p is in, just pos in [0,L] divided by ncells(vector) .INT DANGER.
inline __device__ int3 getCell(float3 p){
  apply_pbc(p); //Reduce to MIC
  // return  int( (p+0.5L)/cellSize )
  return make_int3( p*params.invCellSize + params.getCellFactor ); 
}
inline __device__ int3 getCell(float4 p){
  apply_pbc(p); //Reduce to MIC
  // return  int( (p+0.5L)/cellSize )
  return make_int3( p*params.invCellSize + params.getCellFactor ); 
}

//Apply pbc to a cell coordinates
inline __device__ void pbc_cell(int3 &cell){
  if(cell.x==-1) cell.x = params.xcells-1;
  else if(cell.x==params.xcells) cell.x = 0;

  if(cell.y==-1) cell.y = params.ycells-1;
  else if(cell.y==params.ycells) cell.y = 0;

  if(cell.z==-1) cell.z = params.zcells-1;
  else if(cell.z==params.zcells) cell.z = 0;
}
//Get linear index of a 3D cell, from 0 to ncells-1
inline __device__ uint getCellIndex(int3 gridPos){
  return gridPos.x
    +gridPos.y*params.xcells
    +gridPos.z*params.xcells*params.ycells;
}

/****************************************************************************************/


//Compute the icell of each particle
__global__ void calcCellIndexD(uint *cellIndex, uint *particleIndex, 
			       const float4 __restrict__ *pos, uint N){
  uint index = blockIdx.x*blockDim.x + threadIdx.x;  
  if(index>N) return;
  float4 p = pos[index];

  int3 gridPos = getCell(p);
  int icell = getCellIndex(gridPos);
  /*Before ordering by icell the index in the array is the index!*/
  particleIndex[index] = index;
  cellIndex[index]  = icell;
  
}  
//CPU kernel caller
void calcCellIndex(float4 *pos, uint *cellIndex, uint *particleIndex, uint N){
  calcCellIndexD<<<GPU_Nblocks, GPU_Nthreads>>>(cellIndex, particleIndex, pos, N);
  //cudaCheckErrors("Calc hash");					   
}



//Sort the particleIndex list by cell index,
// this allows to access the neighbour list of each particle fast and coalesced
void sortCellIndex(uint *&cellIndex, uint *&particleIndex, uint N){
  //This uses the CUB API to perform a radix sort
  //CUB orders by key an array pair and copies them onto another pair
  //This function stores an internal key/value pair and switches the arrays each time
   static bool init = false;
   static void *d_temp_storage = NULL;
   static size_t temp_storage_bytes = 0; //Additional storage needed by cub
   static uint *cellIndex_alt = NULL, *particleIndex_alt = NULL; //Additional key/value pair

   static hipcub::DoubleBuffer<uint> d_keys;
   static hipcub::DoubleBuffer<uint> d_values;
   /**Initialize CUB at first call**/
   if(!init){
     /*Allocate temporal value/key pair*/
     gpuErrchk(hipMalloc(&cellIndex_alt, N*sizeof(uint)));
     gpuErrchk(hipMalloc(&particleIndex_alt, N*sizeof(uint)));
     
     /*Create this CUB like data structure*/
     d_keys = hipcub::DoubleBuffer<uint>(cellIndex, cellIndex_alt);    
     d_values = hipcub::DoubleBuffer<uint>(particleIndex, particleIndex_alt);
     /*On first call, this function only computes the size of the required temporal storage*/
     hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
   				    d_keys, 
   				    d_values, N);
     /*Allocate temporary storage*/
     gpuErrchk(hipMalloc(&d_temp_storage, temp_storage_bytes));
     init = true;
   }

   /**Perform the Radix sort on the index/cell pair**/
   hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes,
				   d_keys, 
				   d_values, N); 
   /**Switch the references**/
   cellIndex     = d_keys.Current();
   particleIndex = d_values.Current();

   cellIndex_alt     = d_keys.Alternate();
   particleIndex_alt = d_values.Alternate();

   /*Very important, fix the texture reference!!*/
   gpuErrchk(hipBindTexture(NULL, texParticleIndex,   particleIndex,   (N+1)*sizeof(uint)));

   //Thrust is slower and more memory hungry, for it is a higher level call
   // thrust::sort_by_key(device_ptr<uint>(cellIndex),
   // 		      device_ptr<uint>(cellIndex+N),
   // 		      device_ptr<uint>(particleIndex));

   //cudaCheckErrors("Sort hash");					   
}

//Create CellStart and CellEnd, copy pos onto sortPos
__global__ void reorderAndFindD(float4 *sortPos,
				uint *cellIndex, uint *particleIndex, 
				uint *cellStart, uint *cellEnd,
				float4 *pos,
				uint N){
  uint index = blockIdx.x*blockDim.x + threadIdx.x;
  uint icell, icell2;

  if(index<N){//If my particle is in range
    icell = cellIndex[index]; //Get my icell
    if(index>0)icell2 = cellIndex[index-1];//Get the previous part.'s icell
    else icell2 = 0;
    //If my particle is the first or is in a different cell than the previous
    //my index is the start of a cell
    if(index ==0 || icell != icell2){
      //Then my particle is the first of my cell
      cellStart[icell] = index;
      //If my index is the start of a cell, it is also the end of the previous
      //Except if I am the first one
      if(index>0)
	cellEnd[icell2] = index;
    }
    //If I am the last particle my cell ends 
    if(index == N-1) cellEnd[icell] = index+1;

    //Copy pos into sortPos
    //uint sortIndex   = particleIndex[index];
    uint sortIndex   = tex1Dfetch(texParticleIndex, index);
    sortPos[index]   = pos[sortIndex];
  }

}

//CPU kernel caller
void reorderAndFind(float4 *sortPos,
		    uint *cellIndex, uint *particleIndex, 
		    uint *cellStart, uint *cellEnd, uint ncells,
		    float4 *pos, uint N){
  //Reset CellStart
  hipMemset(cellStart, 0xffffffff, ncells*sizeof(uint));
  //CellEnd does not need reset, a cell with cellStart=0xffffff is not checked for a cellEnd
  reorderAndFindD<<<GPU_Nblocks, GPU_Nthreads>>>(sortPos,
						 cellIndex, particleIndex,
						 cellStart, cellEnd,
						 pos, N);
  //cudaCheckErrors("Reorder and find");					   
}



/***************************************FORCE*****************************/


//Computes the force between to positions
inline __device__ float4 forceij(const float4 &R1,const float4 &R2){

  float3 r12 = make_float3(R2-R1);

  apply_pbc(r12);

  /*Squared distance between 0 and 1*/
  float r2 = dot(r12,r12);
  float r2c = r2*params.invrc2;
  /*Check if i==j. This way reduces warp divergence and its faster than checking i==j outside*/
  //  if(r2c==0.0f) return make_float4(0.0f);  //Both cases handled in texForce
  /*Beyond rcut..*/
  //else if(r2c>=1.0f) return make_float4(0.0f);
  /*Get the force from the texture*/
  float fmod = tex1D(texForce, r2c);
   // float invr2 = 1.0f/r2;
   //  float invr6 = invr2*invr2*invr2;
    //  float invr8 = invr6*invr2;
    //float E =  2.0f*invr6*(invr6-1.0f);
    //float fmod = -48.0f*invr8*invr6+24.0f*invr8;
  return make_float4(fmod*r12);
 }

//Computes the force acting on particle index from particles in cell cell
__device__ float4 forceCell(const int3 &cell, const uint &index,
			    const float4 &pos){
  uint icell  = getCellIndex(cell);
  /*Index of the first particle in the cell's list*/ 
  uint firstParticle = tex1Dfetch(texCellStart, icell);

  float4 force = make_float4(0.0f);
  float4 posj;

  /*Index of the last particle in the cell's list*/
  uint lastParticle = tex1Dfetch(texCellEnd, icell);
  /*Because the list is ordered, all the particle indices in the cell are coalescent!*/
  /*If there are no particles in the cell, firstParticle=0xffffffff, the loop is not computed*/
  /*The fetch of lastParticle eitherway reduces branch divergency and is actually faster than checking
    firstParticle before fetching*/
  for(uint j=firstParticle; j<lastParticle; j++){
    /*Retrieve j pos*/
    posj = tex1Dfetch(texSortPos, j);
    /*Add force, i==j is handled in forceij */
    force += forceij(pos, posj);
  }
   
  return force;
}


//Kernel to compute the force acting on all particles
__global__ void computeForceD(float4* __restrict__ newForce,
			      const uint* __restrict__ particleIndex, 
			      uint N){
  /*Travel the particles per sort order*/
  uint ii =  blockIdx.x*blockDim.x + threadIdx.x;
  
  //Grid-strid loop
  for(int index = ii; index<N; index += blockDim.x * gridDim.x){
    /*Compute force acting on particle particleIndex[index], index in the new order*/
    float4 pos = tex1Dfetch(texSortPos, index);
  
    float4 force = make_float4(0.0f);
    int3 celli = getCell(pos);

    int x,y,z;
    int3 cellj;
    /**Go through all neighbour cells**/
    //For some reason unroll doesnt help here
    for(z=-1; z<=1; z++)
      for(y=-1; y<=1; y++)
	for(x=-1; x<=1; x++){
	  cellj = celli+make_int3(x,y,z);
	  pbc_cell(cellj);	
	  force += forceCell(cellj, index, pos);
	}
    /*Write force with the original order*/
    uint pi = tex1Dfetch(texParticleIndex, index); 
    newForce[pi] += force;
   }
}
__global__ void computeForceDnaive(float4* __restrict__ newForce,
				   const uint* __restrict__ particleIndex, 
				   uint N){
  /*Travel the particles per sort order*/
  uint index =  blockIdx.x*blockDim.x + threadIdx.x;
  if(index>=N) return;
  
  /*Compute force acting on particle particleIndex[index], index in the new order*/
  float4 pos = tex1Dfetch(texSortPos, index);
  float4 posj;
  float4 force = make_float4(0.0f);
  for(int i=0; i<N; i++){
    posj = tex1Dfetch(texSortPos, i);
    force += forceij(pos, posj);
  }

  /*Write force with the original order*/
  uint pi = tex1Dfetch(texParticleIndex, index); 
  newForce[pi] += force;
}

//CPU kernel caller
void computePairForce(float4 *sortPos, float4 *force,
		  uint *cellStart, uint *cellEnd,
		  uint *particleIndex, 
		  uint N){
  computeForceD<<<GPU_Nblocks, GPU_Nthreads>>>(force,
					       particleIndex,
					       N);
  //cudaCheckErrors("computeForce");
}







/****************************ENERGY***************************************/


//Computes the energy between to positions, no cutoff
inline __device__ float energyij(const float4 &R1,const float4 &R2){

  float3 r12 = make_float3(R2-R1);

  apply_pbc(r12);

  float r2 = dot(r12,r12);
  /*Squared distance between 0 and 1*/
  //float r2c = r2*params.invrc2;
  /*Check if i==j. This way reduces warp divergence and its faster than checking i==j outside*/
  if(r2==0.0f) return 0.0f;
  else if(r2*params.invrc2>=1.0f) return 0.0f;
  float invr2 = 1.0f/r2;
  float invr6 = invr2*invr2*invr2;
  float E =  2.0f*invr6*(invr6-1.0f);
  return E;
 }


//Computes the energy acting on particle index from particles in cell cell
__device__ float energyCell(const int3 &cell, const uint &index,
			   const float4 &pos){
  uint icell  = getCellIndex(cell);
  /*Index of the first particle in the cell's list*/ 
  uint firstParticle = tex1Dfetch(texCellStart, icell);

  float energy = 0.0f;
  float4 posj;
  /*Exact copy of forceCell*/
  uint lastParticle = tex1Dfetch(texCellEnd, icell);
  for(uint j=firstParticle; j<lastParticle; j++){
    posj = tex1Dfetch(texSortPos, j);
    energy += energyij(pos, posj);
  }
  return energy;
}




//Kernel to compute the force acting on all particles
__global__ void computeEnergyDnaive(float* __restrict__ Energy,
				    const uint* __restrict__ particleIndex, 
				    uint N){
  /*Travel the particles per sort order*/
  uint index =  blockIdx.x*blockDim.x + threadIdx.x;
  if(index>=N) return;
  
  /*Compute force acting on particle particleIndex[index], index in the new order*/
  float4 pos = tex1Dfetch(texSortPos, index), posj;
  
  float energy = 0.0f;
  //  int3 celli = getCell(pos);

  for(int j=0; j<N; j++){
    posj = tex1Dfetch(texSortPos, j);
    energy += energyij(pos, posj);
  }

  /*Write force with the original order*/
  uint pi = tex1Dfetch(texParticleIndex, index); 
  Energy[pi] = energy;
}
//Kernel to compute the force acting on all particles
__global__ void computeEnergyD(float* __restrict__ Energy,
				    const uint* __restrict__ particleIndex, 
				    uint N){
  /*Travel the particles per sort order*/
  uint index =  blockIdx.x*blockDim.x + threadIdx.x;
  if(index>=N) return;
  
  /*Compute force acting on particle particleIndex[index], index in the new order*/
  float4 pos = tex1Dfetch(texSortPos, index);
  
  float energy = 0.0f;
  int3 celli = getCell(pos);
  int x,y,z;
  int3 cellj;
  /**Go through all neighbour cells**/
  for(z=-1; z<=1; z++)
    for(y=-1; y<=1; y++)
      for(x=-1; x<=1; x++){
	cellj = celli+make_int3(x,y,z);
	pbc_cell(cellj);	
	energy += energyCell(cellj, index, pos);
      }

  /*Write force with the original order*/
  uint pi = tex1Dfetch(texParticleIndex, index); 
  Energy[pi] = energy;
}

//CPU kernel caller
float computePairEnergy(float4 *sortPos, float *energy,
		  uint *cellStart, uint *cellEnd,
		  uint *particleIndex, 
		  uint N){
  computeEnergyD<<<GPU_Nblocks, GPU_Nthreads>>>(energy,
						particleIndex,
   						N);

  device_ptr<float> d_e(energy);
  float sum;
  sum = thrust::reduce(d_e, d_e+N, 0.0f);
  return (sum/(float)N);

  //cudaCheckErrors("computeForce");
}


/****************************VIRIAL***************************************/


//Computes the virial between to positions
inline __device__ float virialij(const float4 &R1,const float4 &R2){

  float3 r12 = make_float3(R2-R1);

  apply_pbc(r12);

  /*Squared distance between 0 and 1*/
  float r2 = dot(r12,r12);
  float r2c = r2*params.invrc2;
  //if(r2c==0.0f) return 0.0f; //No need to check i==j, tex1D(texForce, 0.0) = 0.0
  /*Beyond rcut..*/
  //if(r2c>=1.0f) return 0.0f; //Also 0 in texForce
  /*Get the force from the texture*/
  float fmod = tex1D(texForce, r2c);
  // P = rhoKT + (1/2dV)sum_ij( Fij·rij )
  return dot(fmod*r12,r12);
 }



//Computes the virial acting on particle index from particles in cell cell
__device__ float virialCell(const int3 &cell, const uint &index,
			   const float4 &pos){
  uint icell  = getCellIndex(cell);
  /*Index of the first particle in the cell's list*/ 
  uint firstParticle = tex1Dfetch(texCellStart, icell);

  float virial = 0.0f;
  float4 posj;

  /*Exact copy of forceCell*/
  uint lastParticle = tex1Dfetch(texCellEnd, icell);
  for(uint j=firstParticle; j<lastParticle; j++){
    posj = tex1Dfetch(texSortPos, j);
    virial += virialij(pos, posj);
    }
   
  return virial;
}


//Kernel to compute the force acting on all particles
__global__ void computeVirialDnaive(float* __restrict__ Virial,
				    const uint* __restrict__ particleIndex, 
				    uint N){
  /*Travel the particles per sort order*/
  uint index =  blockIdx.x*blockDim.x + threadIdx.x;
  if(index>=N) return;
  
  /*Compute force acting on particle particleIndex[index], index in the new order*/
  float4 pos = tex1Dfetch(texSortPos, index), posj;
  
  float virial = 0.0f;
  //  int3 celli = getCell(pos);

  for(int j=0; j<N; j++){
    posj = tex1Dfetch(texSortPos, j);
    virial += virialij(pos, posj);
  }

  /*Write force with the original order*/
  uint pi = tex1Dfetch(texParticleIndex, index); 
  Virial[pi] = virial;
}
//Kernel to compute the force acting on all particles
__global__ void computeVirialD(float* __restrict__ Virial,
				    const uint* __restrict__ particleIndex, 
				    uint N){
  /*Travel the particles per sort order*/
  uint index =  blockIdx.x*blockDim.x + threadIdx.x;
  if(index>=N) return;
  
  /*Compute force acting on particle particleIndex[index], index in the new order*/
  float4 pos = tex1Dfetch(texSortPos, index);
  
  float virial = 0.0f;
  int3 celli = getCell(pos);
  int x,y,z;
  int3 cellj;
  /**Go through all neighbour cells**/
  for(z=-1; z<=1; z++)
    for(y=-1; y<=1; y++)
      for(x=-1; x<=1; x++){
	cellj = celli+make_int3(x,y,z);
	pbc_cell(cellj);	
	virial += virialCell(cellj, index, pos);
      }

  /*Write force with the original order*/
  uint pi = tex1Dfetch(texParticleIndex, index); 
  Virial[pi] = virial;
}

//CPU kernel caller
float computePairVirial(float4 *sortPos, float *virial,
		  uint *cellStart, uint *cellEnd,
		  uint *particleIndex, 
		  uint N){
  computeVirialD<<<GPU_Nblocks, GPU_Nthreads>>>(virial,
						particleIndex,
   						N);

  device_ptr<float> d_vir(virial);
  float sum;
  sum = thrust::reduce(d_vir, d_vir+N, 0.0f);
  return (sum/2.0f);

  //cudaCheckErrors("computeForce");
}



/*******************************************DPD********************************************/

/**********************FORCE********************/


//Create CellStart and CellEnd, copy pos onto sortPos
__global__ void reorderAndFindDDPD(float4 *sortPos, float4 *sortVel,
				   uint *cellIndex, uint *particleIndex, 
				   uint *cellStart, uint *cellEnd,
				   const float4 *pos, const float3 *vel,
				   uint N){
  uint index = blockIdx.x*blockDim.x + threadIdx.x;
  uint icell, icell2;

  if(index<N){//If my particle is in range
    icell = cellIndex[index]; //Get my icell
    if(index>0)icell2 = cellIndex[index-1];//Get the previous part.'s icell
    else icell2 = 0;
    //If my particle is the first or is in a different cell than the previous
    //my index is the start of a cell
    if(index ==0 || icell != icell2){
      //Then my particle is the first of my cell
      cellStart[icell] = index;
      //If my index is the start of a cell, it is also the end of the previous
      //Except if I am the first one
      if(index>0)
	cellEnd[icell2] = index;
    }
    //If I am the last particle my cell ends 
    if(index == N-1) cellEnd[icell] = index+1;

    //Copy pos into sortPos
    //uint sortIndex   = particleIndex[index];
    uint sortIndex   = tex1Dfetch(texParticleIndex, index);
    sortPos[index]   = pos[sortIndex];
    sortVel[index]   = make_float4(vel[sortIndex], 0.0f);
  }

}
//CPU kernel caller
void reorderAndFindDPD(float4 *sortPos, float4* sortVel,
		       uint *cellIndex, uint *particleIndex, 
		       uint *cellStart, uint *cellEnd, uint ncells,
		       float4 *pos, float3* vel, uint N){
  //Reset CellStart
  hipMemset(cellStart, 0xffffffff, ncells*sizeof(uint));
  //CellEnd does not need reset, a cell with cellStart=0xffffff is not checked for a cellEnd
  reorderAndFindDDPD<<<GPU_Nblocks, GPU_Nthreads>>>(sortPos, sortVel,
						    cellIndex, particleIndex,
						    cellStart, cellEnd,
						    pos, vel, N);
  //cudaCheckErrors("Reorder and find");					   
}



inline __device__ float randGPU(const ullint &seed, hiprandState *rng){
  hiprand_init(seed, 0, 0, rng);
  return hiprand_normal(rng);
}


//Computes the force between to positions
inline __device__ float4 forceijDPD(const float4 &R1,const float4 &R2,
				    const float4 &V1,const float4 &V2, const float &randij){
  
  float3 r12 = make_float3(R1-R2);
  float3 v12 = make_float3(V1-V2);
  
  apply_pbc(r12);

  float r2 = dot(r12,r12);
  /*Squared distance between 0 and 1*/
  float r2c = r2*params.invrc2;
  
  float fmod= 0.0f;
  
  float w = 0.0f; //The intensity of the DPD thermostat 
  float rinv = 0.0f;
  if(r2c<1.0f){
    if(r2c==0.0f) return make_float4(0.0f);
    //w = r-rc -> linear
    rinv = rsqrt(r2);
    w = rinv-params.invrc;
  }
  else return make_float4(0.0f);
  //fmod = paramsDPD.A*w; //Soft force
  
  fmod -= tex1D(texForce, r2c); //Conservative force
  fmod -= paramsDPD.gamma*w*w*dot(r12,v12); //Damping
  fmod += paramsDPD.noiseAmp*randij*w; //Random force
  return make_float4(fmod*r12);
}

//Computes the force acting on particle index from particles in cell cell
__device__ float4 forceCellDPD(const int3 &cell, const uint &index,
			       const float4 &pos,
			       const float4 &veli,
			       uint N,
			       hiprandState &rng, const ullint &seed){
  uint icell  = getCellIndex(cell);
  /*Index of the first particle in the cell's list*/ 
  uint firstParticle = tex1Dfetch(texCellStart, icell);

  float4 force = make_float4(0.0f);
  float4 posj, velj;
  
  /*Index of the last particle in the cell's list*/
  uint lastParticle = tex1Dfetch(texCellEnd, icell);
  /*Because the list is ordered, all the particle indices in the cell are coalescent!*/
  /*If there are no particles in the cell, firstParticle=0xffffffff, the loop is not computed*/
  /*The fetch of lastParticle eitherway reduces branch divergency and is actually faster than checking
    firstParticle before fetching*/
  float randij;
  ullint i0, j0;
  for(uint j=firstParticle; j<lastParticle; j++){
    /*Retrieve j pos and vel*/
    posj = tex1Dfetch(texSortPos, j);
    velj = tex1Dfetch(texSortVel, j);
    /*Prepare the seed for the RNG, it must be the same seed
      for pair ij and ji!*/
    if(index<j){
      i0=index;
      j0=j;
    }
    else{
      i0=j;
      j0=index;
    }
    /*Get the random number*/
    randij = randGPU(i0+(ullint)N*j0 +seed, &rng);
    /*Sum the force*/
    force += forceijDPD(pos, posj, veli, velj, randij);
  }
   
  return force;
}


//Kernel to compute the force acting on all particles
__global__ void computeForceDDPD(float4* __restrict__ newForce,
				 uint N, ullint seed){
  /*Travel the particles per sort order*/
  uint ii =  blockIdx.x*blockDim.x + threadIdx.x;
  hiprandState rng;
  
  //Grid-stride loop
  for(int index = ii; index<N; index += blockDim.x * gridDim.x){
    uint pi = tex1Dfetch(texParticleIndex, index); 
    /*Compute force acting on particle particleIndex[index], index in the new order*/
    float4 pos = tex1Dfetch(texSortPos, index);
    float4 veli= tex1Dfetch(texSortVel, index);
    //float3 veli = vel[pi];
    float4 force = make_float4(0.0f);
    int3 celli = getCell(pos);

    int x,y,z;
    int3 cellj;
    /**Go through all neighbour cells**/
    //For some reason unroll doesnt help here
    for(z=-1; z<=1; z++)
      for(y=-1; y<=1; y++)
	for(x=-1; x<=1; x++){
	  cellj = celli+make_int3(x,y,z);
	  pbc_cell(cellj);	
	  force += forceCellDPD(cellj, index, pos, veli, N, rng, seed);
	}
    /*Write force with the original order*/
    newForce[pi] += force;
   }
}

//CPU kernel caller
void computePairForceDPD(float4 *force,
			 uint N, ullint seed){
  computeForceDDPD<<<GPU_Nblocks, GPU_Nthreads>>>(force,
						  N, seed);
  //cudaCheckErrors("computeForce");
}





