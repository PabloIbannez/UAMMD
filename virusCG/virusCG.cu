#include "hip/hip_runtime.h"
#include "uammd.cuh"

#include "Integrator/BrownianDynamics.cuh"
#include "Integrator/VerletNVT.cuh"
#include "Integrator/SteepestDescent.cuh"

#include "Interactor/Potential/RadialPotential.cuh"
#include "Interactor/PairForces.cuh"
#include "Interactor/ExternalForces.cuh"
#include "Interactor/NBodyForces.cuh"

#include "third_party/hipcub/hipcub.hpp"

#include <fstream>

using namespace uammd;
using namespace std;

#define GEL_POT

struct forceFunctor{
    ParticleGroup::IndexIterator groupIterator;
    
    real4* force;
    
    forceFunctor(ParticleGroup::IndexIterator groupIterator,real4* force):groupIterator(groupIterator),force(force){}
    
    __host__ __device__ __forceinline__
    real3 operator()(const int &index) const {
    
        int i = groupIterator[index];
        return make_real3(force[i]); 
    }
};

struct virialFunctor{
    int2* pairIterator;
    
    real4* pos;
    real4* force;
    
    virialFunctor(int2* pairIterator,real4* pos,real4* force):pairIterator(pairIterator),pos(pos),force(force){}
    
    __host__ __device__ __forceinline__
    real operator()(const int &index) const {
    
        int i = pairIterator[index].x;
        int j = pairIterator[index].y;
        //return dot(make_real3(force[i]),make_real3(pos[j])); 
        return make_real3(force[i]).z*make_real3(pos[j]).z; 
    }
};

class pressureMeasuring{
    
    private:
        
        shared_ptr<System> sys;
        
        shared_ptr<ParticleData> pd;
        shared_ptr<ParticleGroup> pg;
        
        std::vector<shared_ptr<Interactor>> interactors;
        
        hipStream_t stream;
        
        //
        thrust::device_vector<int2> pairIterator;
        
        //cub reduction variables force
        real3*   totalForce;
        real*    totalVirial;
        void*    cubTempStorageSum = NULL;
        size_t   cubTempStorageSizeSum = 0;
        
        void setUpCubReduction(){
            
            size_t   cubTempStorageSizeForce  = 0;
            size_t   cubTempStorageSizeVirial = 0;
            
            //common
            hipcub::CountingInputIterator<int> countingIterator(0);
            auto groupIterator = pg->getIndexIterator(access::location::gpu);
            
            //force
            auto force = pd->getForce(access::location::gpu, access::mode::read);
            forceFunctor fF(groupIterator,force.raw());
        
            hipcub::TransformInputIterator<real3, forceFunctor, hipcub::CountingInputIterator<int>> forceSumIterator(countingIterator,fF);
            
            hipcub::DeviceReduce::Sum(cubTempStorageSum, cubTempStorageSizeForce, forceSumIterator, totalForce, pg->getNumberParticles(), stream);
            
            //virial
            auto groupIteratorCPU = pg->getIndexIterator(access::location::cpu);
            int i,j;
            for(i = 0  ;i<pg->getNumberParticles();i++){
            for(j = i+1;j<pg->getNumberParticles();j++){
                pairIterator.push_back({groupIteratorCPU[i],groupIteratorCPU[j]});
            }}
            
            int pairNum = pairIterator.size();
            
            auto pos = pd->getPos(access::location::gpu, access::mode::read);
            virialFunctor vF(thrust::raw_pointer_cast(pairIterator.data()),pos.raw(),force.raw());
            
            hipcub::TransformInputIterator<real, virialFunctor, hipcub::CountingInputIterator<int>> viralSumIterator(countingIterator,vF);
            
            hipcub::DeviceReduce::Sum(cubTempStorageSum, cubTempStorageSizeVirial, viralSumIterator, totalVirial, pairNum, stream);
            
            //
            cubTempStorageSizeSum = std::max(cubTempStorageSizeForce,cubTempStorageSizeVirial);
            hipMalloc(&cubTempStorageSum, cubTempStorageSizeSum);
        }
    
    public:
    
        pressureMeasuring(shared_ptr<System> sys,
                          shared_ptr<ParticleData> pd,
                          shared_ptr<ParticleGroup> pg):sys(sys),pd(pd),pg(pg){
          
            sys->log<System::MESSAGE>("[pressureMeasuring] Created.");
            
            hipStreamCreate(&stream);
            
            hipMallocManaged((void**)&totalForce,sizeof(real3));
            hipMallocManaged((void**)&totalVirial,sizeof(real));
            this->setUpCubReduction();
        }
        
        ~pressureMeasuring(){
          
            sys->log<System::MESSAGE>("[pressureMeasuring] Destroyed.");
            
            hipFree(totalForce);
            hipFree(cubTempStorageSum);
            hipStreamDestroy(stream);
        }
        
        void addInteractor(shared_ptr<Interactor> an_interactor){
            interactors.emplace_back(an_interactor);      
        }
    
        real3 sumForce(){
          
            int numberParticles = pg->getNumberParticles();
            auto groupIterator = pg->getIndexIterator(access::location::gpu);
            
            int Nthreads=128;
            int Nblocks=numberParticles/Nthreads + ((numberParticles%Nthreads)?1:0);
            
            {
                auto force = pd->getForce(access::location::gpu, access::mode::readwrite);
                fillWithGPU<<<Nblocks, Nthreads>>>(force.raw(), groupIterator, make_real4(0), numberParticles);
            }
            
            for(auto forceComp: interactors) forceComp->sumForce(stream);
            hipDeviceSynchronize();
            
            hipcub::CountingInputIterator<int> countingIterator(0);
            
            //force
            auto force = pd->getForce(access::location::gpu, access::mode::read);
            forceFunctor fF(groupIterator,force.raw());
            
            hipcub::TransformInputIterator<real3, forceFunctor, hipcub::CountingInputIterator<int>> forceSumIterator(countingIterator,fF);
            
            hipcub::DeviceReduce::Sum(cubTempStorageSum, cubTempStorageSizeSum, forceSumIterator, totalForce, pg->getNumberParticles(), stream);
            hipStreamSynchronize(stream);
            
            return *totalForce;
        }
        
        real sumVirial(){
          
            int pairNum = pairIterator.size();
            int numberParticles = pg->getNumberParticles();
            auto groupIterator = pg->getIndexIterator(access::location::gpu);
            
            int Nthreads=128;
            int Nblocks=pairNum/Nthreads + ((pairNum%Nthreads)?1:0);
            
            {
                auto force = pd->getForce(access::location::gpu, access::mode::readwrite);
                fillWithGPU<<<Nblocks, Nthreads>>>(force.raw(), groupIterator, make_real4(0), numberParticles);
            }
            
            for(auto forceComp: interactors) forceComp->sumForce(stream);
            hipDeviceSynchronize();
            
            hipcub::CountingInputIterator<int> countingIterator(0);
            
            //force
            auto force = pd->getForce(access::location::gpu, access::mode::read);
            auto pos = pd->getPos(access::location::gpu, access::mode::read);
            virialFunctor vF(thrust::raw_pointer_cast(pairIterator.data()),pos.raw(),force.raw());
            
            hipcub::TransformInputIterator<real, virialFunctor, hipcub::CountingInputIterator<int>> viralSumIterator(countingIterator,vF);
            
            hipcub::DeviceReduce::Sum(cubTempStorageSum, cubTempStorageSizeSum, viralSumIterator, totalVirial, pairNum, stream);
            hipStreamSynchronize(stream);
            
            return *totalVirial;
        }
};

double ndOrderEquationPositive(double a,double b,double c){
    return (-b+std::sqrt(b*b-double(4*a*c)))/double(2*a);
}

template<int niter>
double computeRadiusXY(real area,real radiusZ){
    
    double p = 1.6075;
    
    double radiusXY = ndOrderEquationPositive(1.0,2.0*std::pow(radiusZ,p),-3.0*std::pow(area/(4.0*M_PI),p));
           radiusXY = std::pow(radiusXY,1.0/p);
    double epsilon;
    
    //for(int i=0;i<niter;i++){
    //    epsilon  = 1.0-std::pow(radiusXY/radiusZ,int(2));
    //    radiusXY = std::sqrt((1.0/(2.0*M_PI))*(area-M_PI*(radiusZ*radiusZ/epsilon)*std::log((1.0+epsilon)/(1.0-epsilon))));
    //}
    
    return radiusXY;
}

struct Capsid: public ParameterUpdatable{
    
    real epsilon;
    real alphaCut;
    
    real radius2_Z;
    real radius2_XY;
    
    real area;
    
    Capsid(real epsilon, real alphaCut ,real radius):epsilon(epsilon),alphaCut(alphaCut),
                                                     radius2_Z(radius*radius),radius2_XY(radius*radius),area(4.0*M_PI*radius*radius){}

    __device__ __forceinline__ real3 force(const real4 &pos){
        
        real alpha = (pos.x*pos.x+pos.y*pos.y)/(radius2_XY)+
                      pos.z*pos.z/radius2_Z;
        
        if(alpha > alphaCut){
            
            real fmod = (real(1)-alpha)*(real(1)-alpha);
                 fmod = -epsilon/fmod;
                 
            return make_real3(fmod*real(2.0*pos.x/radius2_XY),
                              fmod*real(2.0*pos.y/radius2_XY),
                              fmod*real(2.0*pos.z/radius2_Z));
            
        } else {
            return make_real3(0.0f);
        }
    }
    
    __device__ __forceinline__ real energy(const real4 &pos){
        
        real alpha = (pos.x*pos.x+pos.y*pos.y)/(radius2_XY)+
                      pos.z*pos.z/radius2_Z;
        
        if(alpha > alphaCut){
            
            real energy = real(1)/(real(1)-alpha)-real(1)/(real(1)-alphaCut);
                 energy = epsilon*energy;
                 
            return energy;
            
        } else {
            return real(0);
        }
    }
    
    std::tuple<const real4 *> getArrays(ParticleData *pd){
        auto pos = pd->getPos(access::location::gpu, access::mode::read);
        return std::make_tuple(pos.raw());
    }
    
    real getRadiusZ(){
        return std::sqrt(radius2_Z);
    }
    
    real getRadiusXY(){
        return std::sqrt(radius2_XY);
    }
    
    void setRadiusZ(real newHeight){
        radius2_Z  = newHeight*newHeight;
        radius2_XY = computeRadiusXY<100>(area,newHeight);
        radius2_XY = radius2_XY*radius2_XY;
    }
  
};

struct GEL{
    
    struct InputPairParameters{
        real cutOff;
        real diam;
        real KbT;
        real kappa;
        real B1,B2;
    };
      
    struct PairParameters{
        real cutOff;
        real diam;
        real KbT;
        real kappa;
        real B1,B2;
    };

    static inline __host__ __device__ real force(const real &r2, const PairParameters &params){
        
        real r = sqrt(r2);
        
        if(r >= params.cutOff) return real(0);
        
        //DLVO
        
        real fmod  = (params.KbT*params.B1*exp(-params.kappa*(r-params.diam))/r)*(params.kappa*r+real(1))/r2;
        
        //VdW
             
        real D2divr2   = params.diam*params.diam/r2;
        real D8divr8   = D2divr2*D2divr2*D2divr2*D2divr2;
        real D18divr18 = D8divr8*D8divr8*D2divr2;
        real D36divr36 = D18divr18*D18divr18;
             
             fmod += (params.KbT*params.B2/r2)*(real(36)*D36divr36-real(18)*D18divr18);
             
        return -fmod;      
    }
      
    static inline __host__ __device__ real energy(const real &r2, const PairParameters &params){
        
        real r = sqrt(r2);
        
        if(r >= params.cutOff) return real(0);
        
        //DLVO
        
        real energy  = params.KbT*params.B1*exp(-params.kappa*(r-params.diam))/r;
        
        //VdW
             
        real D2divr2   = params.diam*params.diam/r2;
        real D8divr8   = D2divr2*D2divr2*D2divr2*D2divr2;
        real D18divr18 = D8divr8*D8divr8*D2divr2;
        real D36divr36 = D18divr18*D18divr18;
             
             energy += params.KbT*params.B2*(D36divr36-D18divr18);
        
        return energy;
    }




    static inline __host__ PairParameters processPairParameters(InputPairParameters in_par){

        PairParameters params;
        
        params.cutOff = in_par.cutOff;
        params.diam   = in_par.diam;
        params.KbT    = in_par.KbT;
        params.kappa  = in_par.kappa;
        params.B1     = in_par.B1;
        params.B2     = in_par.B2;
        
        return params;
        
    }
};

struct WCA{
    
    struct InputPairParameters{
        real cutOff;
        real diamEff;
        real epsilon;
    };
      
    struct PairParameters{
        real diamEff;
        real epsilon;
    };

    static inline __host__ __device__ real force(const real &r2, const PairParameters &params){
        
        real r = sqrt(r2);
        
        if(r < real(1.122462)*params.diamEff){
            
            real inv2  = (params.diamEff*params.diamEff)/r2;
            real inv6  = inv2*inv2*inv2;
            real inv12 = inv6*inv6;
            
            real fmod = real(4)*real(6)*params.epsilon*(real(2)*inv12-inv6)/r;
            
            return -fmod;     
        } else {
            return real(0);
        }
    }
      
    static inline __host__ __device__ real energy(const real &r2, const PairParameters &params){
        
        real r = sqrt(r2);
        
        if(r < real(1.122462)*params.diamEff){
            
            real inv2  = (params.diamEff*params.diamEff)/r2;
            real inv6  = inv2*inv2*inv2;
            real inv12 = inv6*inv6;
            
            return real(4)*params.epsilon*(inv12-inv6)+params.epsilon;
        } else {
            return real(0);
        }
    }




    static inline __host__ PairParameters processPairParameters(InputPairParameters in_par){

        PairParameters params;
        
        params.diamEff  = in_par.diamEff;
        params.epsilon  = in_par.epsilon;
        
        return params;
        
    }
};

#ifdef GEL_POT
using potential    = Potential::Radial<GEL>;
#else
using potential    = Potential::Radial<WCA>;
#endif
using pairforces   = PairForces<potential>;
using nbodyforces  = NBodyForces<potential>;

void outputState(shared_ptr<System> sys,shared_ptr<ParticleData> pd,std::ostream& out,real radiusZ, real radiusXY){
    sys->log<System::DEBUG1>("[System] Writing to disk...");
    
    auto pos = pd->getPos(access::location::cpu, access::mode::read).raw();
    auto rad = pd->getRadius(access::location::cpu, access::mode::read).raw();
    
    const int * sortedIndex = pd->getIdOrderedIndices(access::location::cpu);
    
    out<<"#"<<std::endl;
    
    fori(0,pd->getNumParticles()){
        real4 p = pos[sortedIndex[i]];
        real3 pPos = {p.x,p.y,p.z};
        int   type = p.w;
        
        real radius = rad[sortedIndex[i]];
        
        out<<pPos<<" "<<radius<<" "<<type<<endl;
    }
    
    out<<real3({0,0,  radiusZ})<<" "<<1<<" "<<3<<endl;
    out<<real3({0,0, -radiusZ})<<" "<<1<<" "<<3<<endl;
    out<<real3({ radiusXY,0,0})<<" "<<1<<" "<<5<<endl;
    out<<real3({-radiusXY,0,0})<<" "<<1<<" "<<5<<endl;
    out<<real3({0, radiusXY,0})<<" "<<1<<" "<<5<<endl;
    out<<real3({0,-radiusXY,0})<<" "<<1<<" "<<5<<endl;
}

int main(int argc, char *argv[]){
    
    int  N = 200;
    Box  box({100,100,100});
    real V = box.boxSize.x*box.boxSize.y*box.boxSize.z;
    
    real partDiam   =  2;
    real kappa      =  3/partDiam;
    real B1         =  20;
    real B2         =  60;
    
    real cutOff     = partDiam*2.5;
    
    real radius = 15;
    real epsilon  = 1;
    real alphaCut = 0.9;
    
    real h = 0.1;
    real epsilonST = 10;
    real saturationForce     = 100;
    real maxSteps = 100000;
    
    real temperature         = 1;
    real viscosity           = 1;
    real hydrodynamicRadius  = 1;
    real dt                  = 0.01;
    
    int nSteps     = 10000000;
    int printSteps = 10000;
    
    int sortSteps  = 1000;
    
    int decreaseSteps  = 10000;
    
    int measuringSteps = 1000;
    
    ofstream outState("state.sp");
    ofstream outMeasured("measured.dat");
    
    //////////////////////////////////////////////////////////////////////
    
    auto sys = make_shared<System>();
    
    ullint seed = 0xf31337Bada55D00dULL;
    sys->rng().setSeed(seed);
    
    auto pd = make_shared<ParticleData>(N, sys);
    
    ////////////////////////////////////////////////////////////////////
    
    //Initial conditions
    {
        auto pos = pd->getPos(access::location::cpu, access::mode::write).raw();
        auto rad = pd->getRadius(access::location::cpu, access::mode::write).raw();
        
        bool accp = true;
      
        fori(0,N){
            real3 spherePoint  = {real(sys->rng().uniform(-radius,radius)),
                                  real(sys->rng().uniform(-radius,radius)),
                                  real(sys->rng().uniform(-radius,radius))};
            
            real d = spherePoint.x*spherePoint.x+
                     spherePoint.y*spherePoint.y+
                     spherePoint.z*spherePoint.z;
                 
                 d = sqrt(d);
            
            if(d < radius){
                forj(0,i){
                    real3 dst = spherePoint-make_real3(pos[j]);
                    if(sqrt(dot(dst,dst)) < partDiam){
                        accp = false;
                        break;
                    }
                }
                
                if(accp == true){
                    pos[i] = make_real4(spherePoint,1);
                    rad[i] = partDiam*real(0.5);
                } else {
                    i--;
                    accp = true;
                }
            } else {
                i--;
            }
        }
    }
    
    auto pg = make_shared<ParticleGroup>(pd, sys, "All");
    
    ////////////////////////////////////////////////////////////////////
    
    auto pM = make_shared<pressureMeasuring>(sys,pd,pg);
    
    ////////////////////////////////////////////////////////////////////
    
    auto capsidPot   = make_shared<Capsid>(epsilon,alphaCut,radius);
    auto capsidForce = make_shared<ExternalForces<Capsid>>(pd, pg, sys,capsidPot);

    ////////////////////////////////////////////////////////////////////
    
    auto pot = make_shared<potential>(sys);
    
    #ifdef GEL_POT
    
    {
        potential::InputPairParameters params;
        
        params.cutOff = cutOff;
        params.diam   = partDiam;
        params.KbT    = temperature;
        params.kappa  = kappa;
        params.B1     = B1;
        params.B2     = B2;
        
        pot->setPotParameters(0, 0, params);
    }
    
    #else
    
    {
        potential::InputPairParameters params;
        
        params.diamEff   = partDiam;
        params.epsilon = 1;
        
        params.cutOff   = real(1.122462)*partDiam;
        
        pot->setPotParameters(0, 0, params);
    }
    
    #endif
    
    //nbodyforces::Parameters params;
    //params.box = box; 
    //auto partForces = make_shared<nbodyforces>(pd, pg, sys, params, pot);
    
    pairforces::Parameters params;
    params.box = box; 
    auto partForces = make_shared<pairforces>(pd, pg, sys, params, pot);
    
    ////////////////////////////////////////////////////////////////////
    
    //SteepestDescent::Parameters STpar;
    //STpar.h = h;
    //STpar.epsilon = epsilonST;
    //STpar.saturationForce = saturationForce;
    //STpar.maxSteps = maxSteps;
    //
    //auto st = make_shared<SteepestDescent>(pd, pg, sys, STpar);
    //
    //st->addInteractor(capsidForce);
    //st->addInteractor(partForces);
    //
    //outputState(sys,pd,out);
    //
    //while(st->minimizationStep());
    
    ////////////////////////////////////////////////////////////////////
    
    //BD::EulerMaruyama::Parameters parBD;
    //parBD.temperature        = temperature;
    //parBD.viscosity          = viscosity;
    //parBD.hydrodynamicRadius = hydrodynamicRadius;
    //parBD.dt                 = dt;
    //
    //auto bd = make_shared<BD::EulerMaruyama>(pd, pg, sys, parBD); 
    //
    //bd->addInteractor(capsidForce);
    //bd->addInteractor(partForces);
    
    
    VerletNVT::GronbechJensen::Parameters parGJ;
    parGJ.temperature = temperature;
    parGJ.dt = dt;
    parGJ.viscosity = viscosity;
    
    auto gj = make_shared<VerletNVT::GronbechJensen>(pd, pg, sys, parGJ);
    
    gj->addInteractor(capsidForce);
    gj->addInteractor(partForces);
    
    pM->addInteractor(capsidForce);
    pM->addInteractor(partForces);
    
    outputState(sys,pd,outState,capsidPot->getRadiusZ(),capsidPot->getRadiusXY());
    
    ////////////////////////////////////////////////////////////////////
    
    
    Timer tim;
    tim.tic();
    
    sys->log<System::MESSAGE>("SIMULATION STARTS!!!!");
    
    //Run the simulation
    forj(0,nSteps){
        
        //bd->forwardTime();
        gj->forwardTime();
    
        if(j%printSteps ==0) {
            sys->log<System::MESSAGE>("Progress: %.3f%%",100.0*(real(j)/nSteps));
            outputState(sys,pd,outState,capsidPot->getRadiusZ(),capsidPot->getRadiusXY());
        }
        
        if(j%measuringSteps ==0) {
            sys->log<System::MESSAGE>("Measuting ...");
            //outMeasured << capsidPot->getRadiusZ() << " " << pM->sumVirial() << std::endl;
            outMeasured << capsidPot->getRadiusZ() << " " << pM->sumForce().z << std::endl;
        }
        
        if(j%decreaseSteps ==0) {
            sys->log<System::MESSAGE>("Decreasing radius z ...");
            capsidPot->setRadiusZ(capsidPot->getRadiusZ()-0.005);
            sys->log<System::MESSAGE>("New radii: %f (XY) %f (Z)",capsidPot->getRadiusXY(),capsidPot->getRadiusZ());
        }
        
        if(j%sortSteps   == 0){ pd->sortParticles(); }
    }
    
    auto totalTime = tim.toc();
    sys->log<System::MESSAGE>("mean FPS: %.2f", nSteps/totalTime);
    sys->finish();
  
}
